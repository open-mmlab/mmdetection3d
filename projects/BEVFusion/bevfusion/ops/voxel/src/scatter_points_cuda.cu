#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/types.h>

#include <ATen/cuda/CUDAApplyUtils.cuh>

typedef enum { SUM = 0, MEAN = 1, MAX = 2 } reduce_t;

#define CHECK_CUDA(x) \
  TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) \
  TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) \
  CHECK_CUDA(x);       \
  CHECK_CONTIGUOUS(x)

namespace {
int const threadsPerBlock = 512;
int const maxGridDim = 50000;
}  // namespace

__device__ __forceinline__ static void reduceMax(float *address, float val) {
  int *address_as_i = reinterpret_cast<int *>(address);
  int old = *address_as_i, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_i, assumed,
                    __float_as_int(fmaxf(val, __int_as_float(assumed))));
  } while (assumed != old || __int_as_float(old) < val);
}

__device__ __forceinline__ static void reduceMax(double *address, double val) {
  unsigned long long *address_as_ull =
      reinterpret_cast<unsigned long long *>(address);
  unsigned long long old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(
        address_as_ull, assumed,
        __double_as_longlong(fmax(val, __longlong_as_double(assumed))));
  } while (assumed != old || __longlong_as_double(old) < val);
}

// get rid of meaningless warnings when compiling host code
#ifdef __CUDA_ARCH__
__device__ __forceinline__ static void reduceAdd(float *address, float val) {
#if (__CUDA_ARCH__ < 200)
#warning \
    "compute capability lower than 2.x. fall back to use CAS version of atomicAdd for float32"
  int *address_as_i = reinterpret_cast<int *>(address);
  int old = *address_as_i, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_i, assumed,
                    __float_as_int(val + __int_as_float(assumed)));
  } while (assumed != old);
#else
  atomicAdd(address, val);
#endif
}

__device__ __forceinline__ static void reduceAdd(double *address, double val) {
#if (__CUDA_ARCH__ < 600)
#warning \
    "compute capability lower than 6.x. fall back to use CAS version of atomicAdd for float64"
  unsigned long long *address_as_ull =
      reinterpret_cast<unsigned long long *>(address);
  unsigned long long old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
#else
  atomicAdd(address, val);
#endif
}
#endif

template <typename T>
__global__ void
feats_reduce_kernel(const T *feats, const int32_t *coors_map,
                    T *reduced_feats, // shall be 0 at initialization
                    const int num_input, const int num_feats,
                    const reduce_t reduce_type) {
  for (int x = blockIdx.x * blockDim.x + threadIdx.x; x < num_input;
       x += gridDim.x * blockDim.x) {
    int32_t reduce_to = coors_map[x];
    if (reduce_to == -1) continue;

    const T *feats_offset = feats + x * num_feats;
    T *reduced_feats_offset = reduced_feats + reduce_to * num_feats;
    if (reduce_type == reduce_t::MAX) {
      for (int i = 0; i < num_feats; i++) {
        reduceMax(&reduced_feats_offset[i], feats_offset[i]);
      }
    } else {
      for (int i = 0; i < num_feats; i++) {
        reduceAdd(&reduced_feats_offset[i], feats_offset[i]);
      }
    }
  }
}

template <typename T>
__global__ void add_reduce_traceback_grad_kernel(
    T *grad_feats, const T *grad_reduced_feats, const int32_t *coors_map,
    const int32_t *reduce_count, const int num_input, const int num_feats,
    const reduce_t reduce_type) {
  for (int x = blockIdx.x * blockDim.x + threadIdx.x; x < num_input;
       x += gridDim.x * blockDim.x) {
    int32_t reduce_to = coors_map[x];
    if (reduce_to == -1) {
      continue;
    }

    const int input_offset = x * num_feats;
    T *grad_feats_offset = grad_feats + input_offset;
    const int reduced_offset = reduce_to * num_feats;
    const T *grad_reduced_feats_offset = grad_reduced_feats + reduced_offset;

    if (reduce_type == reduce_t::SUM) {
      for (int i = 0; i < num_feats; i++) {
        grad_feats_offset[i] = grad_reduced_feats_offset[i];
      }
    } else if (reduce_type == reduce_t::MEAN) {
      for (int i = 0; i < num_feats; i++) {
        grad_feats_offset[i] = grad_reduced_feats_offset[i] /
                               static_cast<T>(reduce_count[reduce_to]);
      }
    }
  }
}

template <typename T>
__global__ void max_reduce_traceback_scatter_idx_kernel(
    const T *feats, const T *reduced_feats, int32_t *reduce_from,
    const int32_t *coors_map, const int num_input, const int num_feats) {
  for (int x = blockIdx.x * blockDim.x + threadIdx.x; x < num_input;
       x += gridDim.x * blockDim.x) {
    int32_t reduce_to = coors_map[x];

    const int input_offset = x * num_feats;
    const T *feats_offset = feats + input_offset;

    if (reduce_to == -1) {
      continue;
    }

    const int reduced_offset = reduce_to * num_feats;
    const T *reduced_feats_offset = reduced_feats + reduced_offset;
    int32_t *reduce_from_offset = reduce_from + reduced_offset;

    for (int i = 0; i < num_feats; i++) {
      if (feats_offset[i] == reduced_feats_offset[i]) {
        atomicMin(&reduce_from_offset[i], static_cast<int32_t>(x));
      }
    }
  }
}

template <typename T>
__global__ void max_reduce_scatter_grad_kernel(T *grad_feats,
                                               const T *grad_reduced_feats,
                                               const int32_t *reduce_from,
                                               const int num_reduced,
                                               const int num_feats) {
  for (int x = blockIdx.x * blockDim.x + threadIdx.x; x < num_reduced;
       x += gridDim.x * blockDim.x) {
    const int reduced_offset = x * num_feats;
    const int32_t *scatter_to_offset = reduce_from + reduced_offset;
    const T *grad_reduced_feats_offset = grad_reduced_feats + reduced_offset;

    for (int i = 0; i < num_feats; i++) {
      grad_feats[scatter_to_offset[i] * num_feats + i] =
          grad_reduced_feats_offset[i];
    }
  }
}

namespace voxelization {

std::vector<at::Tensor> dynamic_point_to_voxel_forward_gpu(
    const at::Tensor &feats, const at::Tensor &coors,
    const reduce_t reduce_type) {
  CHECK_INPUT(feats);
  CHECK_INPUT(coors);

  const int num_input = feats.size(0);
  const int num_feats = feats.size(1);

  if (num_input == 0)
    return {feats.clone().detach(),
            coors.clone().detach(),
            coors.new_empty({0}, torch::kInt32),
            coors.new_empty({0}, torch::kInt32)};

  at::Tensor out_coors;
  at::Tensor coors_map;
  at::Tensor reduce_count;

  auto coors_clean = coors.masked_fill(coors.lt(0).any(-1, true), -1);

  std::tie(out_coors, coors_map, reduce_count) =
      at::unique_dim(coors_clean, 0, true, true, true);

  if (out_coors.index({0, 0}).lt(0).item<bool>()) {
    // the first element of out_coors (-1,-1,-1) and should be removed
    out_coors = out_coors.slice(0, 1);
    reduce_count = reduce_count.slice(0, 1);
    coors_map = coors_map - 1;
  }

  coors_map = coors_map.to(torch::kInt32);
  reduce_count = reduce_count.to(torch::kInt32);

  auto reduced_feats =
      at::empty({out_coors.size(0), num_feats}, feats.options());

  AT_DISPATCH_FLOATING_TYPES(
      feats.scalar_type(), "feats_reduce_kernel", ([&] {
    if (reduce_type == reduce_t::MAX)
      reduced_feats.fill_(-std::numeric_limits<scalar_t>::infinity());
    else
      reduced_feats.fill_(static_cast<scalar_t>(0));

    dim3 blocks(std::min(at::cuda::ATenCeilDiv(num_input, threadsPerBlock),
                         maxGridDim));
    dim3 threads(threadsPerBlock);
    feats_reduce_kernel<<<blocks, threads>>>(
        feats.data_ptr<scalar_t>(), coors_map.data_ptr<int32_t>(),
        reduced_feats.data_ptr<scalar_t>(), num_input, num_feats, reduce_type);
    if (reduce_type == reduce_t::MEAN)
      reduced_feats /= reduce_count.unsqueeze(-1).to(reduced_feats.dtype());
  }));
  AT_CUDA_CHECK(hipGetLastError());

  return {reduced_feats, out_coors, coors_map, reduce_count};
}

void dynamic_point_to_voxel_backward_gpu(at::Tensor &grad_feats,
                                         const at::Tensor &grad_reduced_feats,
                                         const at::Tensor &feats,
                                         const at::Tensor &reduced_feats,
                                         const at::Tensor &coors_map,
                                         const at::Tensor &reduce_count,
                                         const reduce_t reduce_type) {
  CHECK_INPUT(grad_feats);
  CHECK_INPUT(grad_reduced_feats);
  CHECK_INPUT(feats);
  CHECK_INPUT(reduced_feats);
  CHECK_INPUT(coors_map);
  CHECK_INPUT(reduce_count);

  const int num_input = feats.size(0);
  const int num_reduced = reduced_feats.size(0);
  const int num_feats = feats.size(1);

  grad_feats.fill_(0);
  // copy voxel grad to points

  if (num_input == 0 || num_reduced == 0) return;

  if (reduce_type == reduce_t::MEAN || reduce_type == reduce_t::SUM) {
    AT_DISPATCH_FLOATING_TYPES(
        grad_reduced_feats.scalar_type(), "add_reduce_traceback_grad_kernel",
        ([&] {
          dim3 blocks(std::min(
              at::cuda::ATenCeilDiv(num_input, threadsPerBlock), maxGridDim));
          dim3 threads(threadsPerBlock);
          add_reduce_traceback_grad_kernel<<<blocks, threads>>>(
              grad_feats.data_ptr<scalar_t>(),
              grad_reduced_feats.data_ptr<scalar_t>(),
              coors_map.data_ptr<int32_t>(), reduce_count.data_ptr<int32_t>(),
              num_input, num_feats, reduce_type);
        }));
    AT_CUDA_CHECK(hipGetLastError());
  } else {
    auto reduce_from = at::full({num_reduced, num_feats}, num_input,
                                coors_map.options().dtype(torch::kInt32));
    AT_DISPATCH_FLOATING_TYPES(
        grad_reduced_feats.scalar_type(),
        "max_reduce_traceback_scatter_idx_kernel", ([&] {
          dim3 blocks(std::min(
              at::cuda::ATenCeilDiv(num_input, threadsPerBlock), maxGridDim));
          dim3 threads(threadsPerBlock);
          max_reduce_traceback_scatter_idx_kernel<<<blocks, threads>>>(
              feats.data_ptr<scalar_t>(), reduced_feats.data_ptr<scalar_t>(),
              reduce_from.data_ptr<int32_t>(), coors_map.data_ptr<int32_t>(),
              num_input, num_feats);
        }));
    AT_CUDA_CHECK(hipGetLastError());

    AT_DISPATCH_FLOATING_TYPES(
        grad_reduced_feats.scalar_type(),
        "max_reduce_traceback_scatter_idx_kernel", ([&] {
          dim3 blocks(std::min(
              at::cuda::ATenCeilDiv(num_reduced, threadsPerBlock), maxGridDim));
          dim3 threads(threadsPerBlock);
          max_reduce_scatter_grad_kernel<<<blocks, threads>>>(
              grad_feats.data_ptr<scalar_t>(),
              grad_reduced_feats.data_ptr<scalar_t>(),
              reduce_from.data_ptr<int32_t>(), num_reduced, num_feats);
        }));
    AT_CUDA_CHECK(hipGetLastError());
  }
  return;
}

}  // namespace voxelization
