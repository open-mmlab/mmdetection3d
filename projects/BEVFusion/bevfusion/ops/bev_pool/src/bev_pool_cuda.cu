
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/*
  Function: pillar pooling
  Args:
    b                : batch size
    d                : depth of the feature map
    h                : height of pooled feature map
    w                : width of pooled feature map
    n                : number of input points
    c                : number of channels
    n_intervals      : number of unique points
    x                : input features, FloatTensor[n, c]
    geom_feats       : input coordinates, IntTensor[n, 4]
    interval_lengths : starting position for pooled point, IntTensor[n_intervals]
    interval_starts  : how many points in each pooled point, IntTensor[n_intervals]
    out              : output features, FloatTensor[b, d, h, w, c]
*/
__global__ void bev_pool_kernel(int b, int d, int h, int w, int n, int c, int n_intervals,
                                  const float *__restrict__ x,
                                  const int *__restrict__ geom_feats,
                                  const int *__restrict__ interval_starts,
                                  const int *__restrict__ interval_lengths,
                                  float* __restrict__ out) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int index = idx / c;
  int cur_c = idx % c;
  if (index >= n_intervals) return;
  int interval_start = interval_starts[index];
  int interval_length = interval_lengths[index];
  const int* cur_geom_feats = geom_feats + interval_start * 4;
  const float* cur_x = x + interval_start * c + cur_c;
  float* cur_out = out + cur_geom_feats[3] * d * h * w * c +
    cur_geom_feats[2] * h * w * c + cur_geom_feats[0] * w * c +
    cur_geom_feats[1] * c + cur_c;
  float psum = 0;
  for(int i = 0; i < interval_length; i++){
    psum += cur_x[i * c];
  }
  *cur_out = psum;
}


/*
  Function: pillar pooling backward
  Args:
    b                : batch size
    d                : depth of the feature map
    h                : height of pooled feature map
    w                : width of pooled feature map
    n                : number of input points
    c                : number of channels
    n_intervals      : number of unique points
    out_grad         : gradient of the BEV fmap from top, FloatTensor[b, d, h, w, c]
    geom_feats       : input coordinates, IntTensor[n, 4]
    interval_lengths : starting position for pooled point, IntTensor[n_intervals]
    interval_starts  : how many points in each pooled point, IntTensor[n_intervals]
    x_grad           : gradient of the image fmap, FloatTensor
*/
__global__ void bev_pool_grad_kernel(int b, int d, int h, int w, int n, int c, int n_intervals,
                                  const float *__restrict__ out_grad,
                                  const int *__restrict__ geom_feats,
                                  const int *__restrict__ interval_starts,
                                  const int *__restrict__ interval_lengths,
                                  float* __restrict__ x_grad) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int index = idx / c;
  int cur_c = idx % c;
  if (index >= n_intervals) return;
  int interval_start = interval_starts[index];
  int interval_length = interval_lengths[index];

  const int* cur_geom_feats = geom_feats + interval_start * 4;
  float* cur_x_grad = x_grad + interval_start * c + cur_c;

  const float* cur_out_grad = out_grad + cur_geom_feats[3] * d * h * w * c +
    cur_geom_feats[2] * h * w * c + cur_geom_feats[0] * w * c +
    cur_geom_feats[1] * c + cur_c;
  for(int i = 0; i < interval_length; i++){
    cur_x_grad[i * c] = *cur_out_grad;
  }

}

void bev_pool(int b, int d, int h, int w, int n, int c, int n_intervals, const float* x,
  const int* geom_feats, const int* interval_starts, const int* interval_lengths, float* out) {
  bev_pool_kernel<<<(int)ceil(((double)n_intervals * c / 256)), 256>>>(
    b, d, h, w, n, c, n_intervals, x, geom_feats, interval_starts, interval_lengths, out
  );
}

void bev_pool_grad(int b, int d, int h, int w, int n, int c, int n_intervals, const float* out_grad,
  const int* geom_feats, const int* interval_starts, const int* interval_lengths, float* x_grad) {
  bev_pool_grad_kernel<<<(int)ceil(((double)n_intervals * c / 256)), 256>>>(
    b, d, h, w, n, c, n_intervals, out_grad, geom_feats, interval_starts, interval_lengths, x_grad
  );
}
