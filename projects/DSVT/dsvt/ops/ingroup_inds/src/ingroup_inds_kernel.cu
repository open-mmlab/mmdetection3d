#include "hip/hip_runtime.h"
#include <assert.h>
#include <vector>
#include <math.h>
#include <stdio.h>
#include <torch/serialize/tensor.h>
#include <torch/extension.h>
#include <torch/types.h>
#include "hip/hip_fp16.h"
// #include "error.cuh"

#define CHECK_CALL(call)                                   \
do                                                    \
{                                                     \
    const hipError_t error_code = call;              \
    if (error_code != hipSuccess)                    \
    {                                                 \
        printf("CUDA Error:\n");                      \
        printf("    File:       %s\n", __FILE__);     \
        printf("    Line:       %d\n", __LINE__);     \
        printf("    Error code: %d\n", error_code);   \
        printf("    Error text: %s\n",                \
            hipGetErrorString(error_code));          \
        exit(1);                                      \
    }                                                 \
} while (0)

#define THREADS_PER_BLOCK 256
#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))

// #define DEBUG
// #define ASSERTION

__global__ void ingroup_inds_kernel(
    const long *group_inds,
    long *out_inds,
    int *ingroup_counter,
    int N
) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N) return;
  long this_group_id = group_inds[idx];

  int cnt = atomicAdd(&ingroup_counter[this_group_id], 1);
  out_inds[idx] = cnt;
}


 void ingroup_inds_launcher(
  const long *group_inds,
  long *out_inds,
  int N,
  int max_group_id
  ) {

  int *ingroup_counter = NULL;
  CHECK_CALL(hipMalloc(&ingroup_counter,   (max_group_id + 1) * sizeof(int)));
  CHECK_CALL(hipMemset(ingroup_counter, 0, (max_group_id + 1) * sizeof(int)));

  dim3 blocks(DIVUP(N, THREADS_PER_BLOCK));
  dim3 threads(THREADS_PER_BLOCK);

  ingroup_inds_kernel<<<blocks, threads>>>(
      group_inds,
      out_inds,
      ingroup_counter,
      N
  );

  hipFree(ingroup_counter);

  #ifdef DEBUG
  CHECK_CALL(hipGetLastError());
  CHECK_CALL(hipDeviceSynchronize());
  #endif

  return;

}
