#include "hip/hip_runtime.h"
// Modified from
// https://github.com/sshaoshuai/PCDet/blob/master/pcdet/ops/roiaware_pool3d/src/roiaware_pool3d_kernel.cu
// Written by Shaoshuai Shi
// All Rights Reserved 2019.

#include <assert.h>
#include <math.h>
#include <stdio.h>
#include <torch/serialize/tensor.h>
#include <torch/types.h>

#define THREADS_PER_BLOCK 256
#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))

// #define DEBUG

__device__ inline void lidar_to_local_coords(float shift_x, float shift_y,
                                             float rz, float &local_x,
                                             float &local_y) {
  // should rotate pi/2 + alpha to translate LiDAR to local
  float rot_angle = rz + M_PI / 2;
  float cosa = cos(rot_angle), sina = sin(rot_angle);
  local_x = shift_x * cosa + shift_y * (-sina);
  local_y = shift_x * sina + shift_y * cosa;
}

__device__ inline int check_pt_in_box3d(const float *pt, const float *box3d,
                                        float &local_x, float &local_y) {
  // param pt: (x, y, z)
  // param box3d: (cx, cy, cz, w, l, h, rz) in LiDAR coordinate, cz in the
  // bottom center
  float x = pt[0], y = pt[1], z = pt[2];
  float cx = box3d[0], cy = box3d[1], cz = box3d[2];
  float w = box3d[3], l = box3d[4], h = box3d[5], rz = box3d[6];
  cz += h / 2.0;  // shift to the center since cz in box3d is the bottom center

  if (fabsf(z - cz) > h / 2.0) return 0;
  lidar_to_local_coords(x - cx, y - cy, rz, local_x, local_y);
  float in_flag = (local_x > -l / 2.0) & (local_x < l / 2.0) &
                  (local_y > -w / 2.0) & (local_y < w / 2.0);
  return in_flag;
}

__global__ void generate_pts_mask_for_box3d(int boxes_num, int pts_num,
                                            int out_x, int out_y, int out_z,
                                            const float *rois, const float *pts,
                                            int *pts_mask) {
  // params rois: (N, 7) [x, y, z, w, l, h, rz] in LiDAR coordinate
  // params pts: (npoints, 3) [x, y, z]
  // params pts_mask: (N, npoints): -1 means point doesnot in this box,
  // otherwise: encode (x_idxs, y_idxs, z_idxs) by binary bit
  int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int box_idx = blockIdx.y;
  if (pt_idx >= pts_num || box_idx >= boxes_num) return;

  pts += pt_idx * 3;
  rois += box_idx * 7;
  pts_mask += box_idx * pts_num + pt_idx;

  float local_x = 0, local_y = 0;
  int cur_in_flag = check_pt_in_box3d(pts, rois, local_x, local_y);

  pts_mask[0] = -1;
  if (cur_in_flag > 0) {
    float local_z = pts[2] - rois[2];
    float w = rois[3], l = rois[4], h = rois[5];

    float x_res = l / out_x;
    float y_res = w / out_y;
    float z_res = h / out_z;

    unsigned int x_idx = int((local_x + l / 2) / x_res);
    unsigned int y_idx = int((local_y + w / 2) / y_res);
    unsigned int z_idx = int(local_z / z_res);

    x_idx = min(max(x_idx, 0), out_x - 1);
    y_idx = min(max(y_idx, 0), out_y - 1);
    z_idx = min(max(z_idx, 0), out_z - 1);

    unsigned int idx_encoding = (x_idx << 16) + (y_idx << 8) + z_idx;
#ifdef DEBUG
    printf(
        "mask: pts_%d(%.3f, %.3f, %.3f), local(%.3f, %.3f, %.3f), idx(%d, %d, "
        "%d), res(%.3f, %.3f, %.3f), idx_encoding=%x\n",
        pt_idx, pts[0], pts[1], pts[2], local_x, local_y, local_z, x_idx, y_idx,
        z_idx, x_res, y_res, z_res, idx_encoding);
#endif

    pts_mask[0] = idx_encoding;
  }
}

__global__ void collect_inside_pts_for_box3d(int boxes_num, int pts_num,
                                             int max_pts_each_voxel, int out_x,
                                             int out_y, int out_z,
                                             const int *pts_mask,
                                             int *pts_idx_of_voxels) {
  // params pts_mask: (N, npoints)  0 or 1
  // params pts_idx_of_voxels: (N, out_x, out_y, out_z, max_pts_each_voxel)

  int box_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (box_idx >= boxes_num) return;

  int max_num_pts = max_pts_each_voxel - 1;  // index 0 is the counter
  pts_idx_of_voxels += box_idx * out_x * out_y * out_z * max_pts_each_voxel;

  for (int k = 0; k < pts_num; k++) {
    if (pts_mask[box_idx * pts_num + k] != -1) {
      unsigned int idx_encoding = pts_mask[box_idx * pts_num + k];
      unsigned int x_idx = (idx_encoding >> 16) & 0xFF;
      unsigned int y_idx = (idx_encoding >> 8) & 0xFF;
      unsigned int z_idx = idx_encoding & 0xFF;
      unsigned int base_offset = x_idx * out_y * out_z * max_pts_each_voxel +
                                 y_idx * out_z * max_pts_each_voxel +
                                 z_idx * max_pts_each_voxel;
      unsigned int cnt = pts_idx_of_voxels[base_offset];
      if (cnt < max_num_pts) {
        pts_idx_of_voxels[base_offset + cnt + 1] = k;
        pts_idx_of_voxels[base_offset]++;
      }
#ifdef DEBUG
      printf("collect: pts_%d, idx(%d, %d, %d), idx_encoding=%x\n", k, x_idx,
             y_idx, z_idx, idx_encoding);
#endif
    }
  }
}

__global__ void roiaware_maxpool3d(int boxes_num, int pts_num, int channels,
                                   int max_pts_each_voxel, int out_x, int out_y,
                                   int out_z, const float *pts_feature,
                                   const int *pts_idx_of_voxels,
                                   float *pooled_features, int *argmax) {
  // params pts_feature: (npoints, C)
  // params pts_idx_of_voxels: (N, out_x, out_y, out_z, max_pts_each_voxel),
  // index 0 is the counter params pooled_features: (N, out_x, out_y, out_z, C)
  // params argmax: (N, out_x, out_y, out_z, C)

  int box_idx = blockIdx.z;
  int channel_idx = blockIdx.y;
  int voxel_idx_flat = blockIdx.x * blockDim.x + threadIdx.x;

  int x_idx = voxel_idx_flat / (out_y * out_z);
  int y_idx = (voxel_idx_flat - x_idx * (out_y * out_z)) / out_z;
  int z_idx = voxel_idx_flat % out_z;
  if (box_idx >= boxes_num || channel_idx >= channels || x_idx >= out_x ||
      y_idx >= out_y || z_idx >= out_z)
    return;

#ifdef DEBUG
  printf("src pts_idx_of_voxels: (%p, ), argmax: %p\n", pts_idx_of_voxels,
         argmax);
#endif

  int offset_base = x_idx * out_y * out_z + y_idx * out_z + z_idx;
  pts_idx_of_voxels += box_idx * out_x * out_y * out_z * max_pts_each_voxel +
                       offset_base * max_pts_each_voxel;
  pooled_features += box_idx * out_x * out_y * out_z * channels +
                     offset_base * channels + channel_idx;
  argmax += box_idx * out_x * out_y * out_z * channels +
            offset_base * channels + channel_idx;

  int argmax_idx = -1;
  float max_val = -1e50;

  int total_pts = pts_idx_of_voxels[0];

  for (int k = 1; k <= total_pts; k++) {
    if (pts_feature[pts_idx_of_voxels[k] * channels + channel_idx] > max_val) {
      max_val = pts_feature[pts_idx_of_voxels[k] * channels + channel_idx];
      argmax_idx = pts_idx_of_voxels[k];
    }
  }

  if (argmax_idx != -1) {
    pooled_features[0] = max_val;
  }
  argmax[0] = argmax_idx;

#ifdef DEBUG
  printf(
      "channel_%d idx(%d, %d, %d), argmax_idx=(%d, %.3f), total=%d, after "
      "pts_idx: %p, argmax: (%p, %d)\n",
      channel_idx, x_idx, y_idx, z_idx, argmax_idx, max_val, total_pts,
      pts_idx_of_voxels, argmax, argmax_idx);
#endif
}

__global__ void roiaware_avgpool3d(int boxes_num, int pts_num, int channels,
                                   int max_pts_each_voxel, int out_x, int out_y,
                                   int out_z, const float *pts_feature,
                                   const int *pts_idx_of_voxels,
                                   float *pooled_features) {
  // params pts_feature: (npoints, C)
  // params pts_idx_of_voxels: (N, out_x, out_y, out_z, max_pts_each_voxel),
  // index 0 is the counter params pooled_features: (N, out_x, out_y, out_z, C)
  // params argmax: (N, out_x, out_y, out_z, C)

  int box_idx = blockIdx.z;
  int channel_idx = blockIdx.y;
  int voxel_idx_flat = blockIdx.x * blockDim.x + threadIdx.x;

  int x_idx = voxel_idx_flat / (out_y * out_z);
  int y_idx = (voxel_idx_flat - x_idx * (out_y * out_z)) / out_z;
  int z_idx = voxel_idx_flat % out_z;
  if (box_idx >= boxes_num || channel_idx >= channels || x_idx >= out_x ||
      y_idx >= out_y || z_idx >= out_z)
    return;

  int offset_base = x_idx * out_y * out_z + y_idx * out_z + z_idx;
  pts_idx_of_voxels += box_idx * out_x * out_y * out_z * max_pts_each_voxel +
                       offset_base * max_pts_each_voxel;
  pooled_features += box_idx * out_x * out_y * out_z * channels +
                     offset_base * channels + channel_idx;

  float sum_val = 0;
  int total_pts = pts_idx_of_voxels[0];

  for (int k = 1; k <= total_pts; k++) {
    sum_val += pts_feature[pts_idx_of_voxels[k] * channels + channel_idx];
  }

  if (total_pts > 0) {
    pooled_features[0] = sum_val / total_pts;
  }
}

void roiaware_pool3d_launcher(int boxes_num, int pts_num, int channels,
                              int max_pts_each_voxel, int out_x, int out_y,
                              int out_z, const float *rois, const float *pts,
                              const float *pts_feature, int *argmax,
                              int *pts_idx_of_voxels, float *pooled_features,
                              int pool_method) {
  // params rois: (N, 7) [x, y, z, w, l, h, rz] in LiDAR coordinate
  // params pts: (npoints, 3) [x, y, z] in LiDAR coordinate
  // params pts_feature: (npoints, C)
  // params argmax: (N, out_x, out_y, out_z, C)
  // params pts_idx_of_voxels: (N, out_x, out_y, out_z, max_pts_each_voxel)
  // params pooled_features: (N, out_x, out_y, out_z, C)
  // params pool_method: 0: max_pool 1: avg_pool

  int *pts_mask = NULL;
  hipMalloc(&pts_mask, boxes_num * pts_num * sizeof(int));  // (N, M)
  hipMemset(pts_mask, -1, boxes_num * pts_num * sizeof(int));

  dim3 blocks_mask(DIVUP(pts_num, THREADS_PER_BLOCK), boxes_num);
  dim3 threads(THREADS_PER_BLOCK);
  generate_pts_mask_for_box3d<<<blocks_mask, threads>>>(
      boxes_num, pts_num, out_x, out_y, out_z, rois, pts, pts_mask);

  // TODO: Merge the collect and pool functions, SS

  dim3 blocks_collect(DIVUP(boxes_num, THREADS_PER_BLOCK));
  collect_inside_pts_for_box3d<<<blocks_collect, threads>>>(
      boxes_num, pts_num, max_pts_each_voxel, out_x, out_y, out_z, pts_mask,
      pts_idx_of_voxels);

  dim3 blocks_pool(DIVUP(out_x * out_y * out_z, THREADS_PER_BLOCK), channels,
                   boxes_num);
  if (pool_method == 0) {
    roiaware_maxpool3d<<<blocks_pool, threads>>>(
        boxes_num, pts_num, channels, max_pts_each_voxel, out_x, out_y, out_z,
        pts_feature, pts_idx_of_voxels, pooled_features, argmax);
  } else if (pool_method == 1) {
    roiaware_avgpool3d<<<blocks_pool, threads>>>(
        boxes_num, pts_num, channels, max_pts_each_voxel, out_x, out_y, out_z,
        pts_feature, pts_idx_of_voxels, pooled_features);
  }

  hipFree(pts_mask);

#ifdef DEBUG
  hipDeviceSynchronize();  // for using printf in kernel function
#endif
}

__global__ void roiaware_maxpool3d_backward(int boxes_num, int channels,
                                            int out_x, int out_y, int out_z,
                                            const int *argmax,
                                            const float *grad_out,
                                            float *grad_in) {
  // params argmax: (N, out_x, out_y, out_z, C)
  // params grad_out: (N, out_x, out_y, out_z, C)
  // params grad_in: (npoints, C), return value

  int box_idx = blockIdx.z;
  int channel_idx = blockIdx.y;
  int voxel_idx_flat = blockIdx.x * blockDim.x + threadIdx.x;

  int x_idx = voxel_idx_flat / (out_y * out_z);
  int y_idx = (voxel_idx_flat - x_idx * (out_y * out_z)) / out_z;
  int z_idx = voxel_idx_flat % out_z;
  if (box_idx >= boxes_num || channel_idx >= channels || x_idx >= out_x ||
      y_idx >= out_y || z_idx >= out_z)
    return;

  int offset_base = x_idx * out_y * out_z + y_idx * out_z + z_idx;
  argmax += box_idx * out_x * out_y * out_z * channels +
            offset_base * channels + channel_idx;
  grad_out += box_idx * out_x * out_y * out_z * channels +
              offset_base * channels + channel_idx;

  if (argmax[0] == -1) return;

  atomicAdd(grad_in + argmax[0] * channels + channel_idx, grad_out[0] * 1);
}

__global__ void roiaware_avgpool3d_backward(int boxes_num, int channels,
                                            int out_x, int out_y, int out_z,
                                            int max_pts_each_voxel,
                                            const int *pts_idx_of_voxels,
                                            const float *grad_out,
                                            float *grad_in) {
  // params pts_idx_of_voxels: (N, out_x, out_y, out_z, max_pts_each_voxel)
  // params grad_out: (N, out_x, out_y, out_z, C)
  // params grad_in: (npoints, C), return value

  int box_idx = blockIdx.z;
  int channel_idx = blockIdx.y;
  int voxel_idx_flat = blockIdx.x * blockDim.x + threadIdx.x;

  int x_idx = voxel_idx_flat / (out_y * out_z);
  int y_idx = (voxel_idx_flat - x_idx * (out_y * out_z)) / out_z;
  int z_idx = voxel_idx_flat % out_z;
  if (box_idx >= boxes_num || channel_idx >= channels || x_idx >= out_x ||
      y_idx >= out_y || z_idx >= out_z)
    return;

  int offset_base = x_idx * out_y * out_z + y_idx * out_z + z_idx;
  pts_idx_of_voxels += box_idx * out_x * out_y * out_z * max_pts_each_voxel +
                       offset_base * max_pts_each_voxel;
  grad_out += box_idx * out_x * out_y * out_z * channels +
              offset_base * channels + channel_idx;

  int total_pts = pts_idx_of_voxels[0];
  float cur_grad = 1 / fmaxf(float(total_pts), 1.0);
  for (int k = 1; k <= total_pts; k++) {
    atomicAdd(grad_in + pts_idx_of_voxels[k] * channels + channel_idx,
              grad_out[0] * cur_grad);
  }
}

void roiaware_pool3d_backward_launcher(int boxes_num, int out_x, int out_y,
                                       int out_z, int channels,
                                       int max_pts_each_voxel,
                                       const int *pts_idx_of_voxels,
                                       const int *argmax, const float *grad_out,
                                       float *grad_in, int pool_method) {
  // params pts_idx_of_voxels: (N, out_x, out_y, out_z, max_pts_each_voxel)
  // params argmax: (N, out_x, out_y, out_z, C)
  // params grad_out: (N, out_x, out_y, out_z, C)
  // params grad_in: (npoints, C), return value
  // params pool_method: 0: max_pool, 1: avg_pool

  dim3 blocks(DIVUP(out_x * out_y * out_z, THREADS_PER_BLOCK), channels,
              boxes_num);
  dim3 threads(THREADS_PER_BLOCK);
  if (pool_method == 0) {
    roiaware_maxpool3d_backward<<<blocks, threads>>>(
        boxes_num, channels, out_x, out_y, out_z, argmax, grad_out, grad_in);
  } else if (pool_method == 1) {
    roiaware_avgpool3d_backward<<<blocks, threads>>>(
        boxes_num, channels, out_x, out_y, out_z, max_pts_each_voxel,
        pts_idx_of_voxels, grad_out, grad_in);
  }
}
