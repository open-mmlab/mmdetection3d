// Modified from
// https://github.com/sshaoshuai/Pointnet2.PyTorch/tree/master/pointnet2/src/group_points_gpu.cu


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define THREADS_PER_BLOCK 256
#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))

__global__ void group_points_grad_kernel(int b, int c, int n, int npoints,
                                         int nsample,
                                         const float *__restrict__ grad_out,
                                         const int *__restrict__ idx,
                                         float *__restrict__ grad_points) {
  // grad_out: (B, C, npoints, nsample)
  // idx: (B, npoints, nsample)
  // output:
  //      grad_points: (B, C, N)
  int bs_idx = blockIdx.z;
  int c_idx = blockIdx.y;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int pt_idx = index / nsample;
  if (bs_idx >= b || c_idx >= c || pt_idx >= npoints) return;

  int sample_idx = index % nsample;
  grad_out += bs_idx * c * npoints * nsample + c_idx * npoints * nsample +
              pt_idx * nsample + sample_idx;
  idx += bs_idx * npoints * nsample + pt_idx * nsample + sample_idx;

  atomicAdd(grad_points + bs_idx * c * n + c_idx * n + idx[0], grad_out[0]);
}

void group_points_grad_kernel_launcher(int b, int c, int n, int npoints,
                                       int nsample, const float *grad_out,
                                       const int *idx, float *grad_points,
                                       hipStream_t stream) {
  // grad_out: (B, C, npoints, nsample)
  // idx: (B, npoints, nsample)
  // output:
  //      grad_points: (B, C, N)
  hipError_t err;
  dim3 blocks(DIVUP(npoints * nsample, THREADS_PER_BLOCK), c,
              b);  // blockIdx.x(col), blockIdx.y(row)
  dim3 threads(THREADS_PER_BLOCK);

  group_points_grad_kernel<<<blocks, threads, 0, stream>>>(
      b, c, n, npoints, nsample, grad_out, idx, grad_points);

  err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
    exit(-1);
  }
}

__global__ void group_points_kernel(int b, int c, int n, int npoints,
                                    int nsample,
                                    const float *__restrict__ points,
                                    const int *__restrict__ idx,
                                    float *__restrict__ out) {
  // points: (B, C, N)
  // idx: (B, npoints, nsample)
  // output:
  //      out: (B, C, npoints, nsample)
  int bs_idx = blockIdx.z;
  int c_idx = blockIdx.y;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int pt_idx = index / nsample;
  if (bs_idx >= b || c_idx >= c || pt_idx >= npoints) return;

  int sample_idx = index % nsample;

  idx += bs_idx * npoints * nsample + pt_idx * nsample + sample_idx;
  int in_idx = bs_idx * c * n + c_idx * n + idx[0];
  int out_idx = bs_idx * c * npoints * nsample + c_idx * npoints * nsample +
                pt_idx * nsample + sample_idx;

  out[out_idx] = points[in_idx];
}

void group_points_kernel_launcher(int b, int c, int n, int npoints, int nsample,
                                  const float *points, const int *idx,
                                  float *out, hipStream_t stream) {
  // points: (B, C, N)
  // idx: (B, npoints, nsample)
  // output:
  //      out: (B, C, npoints, nsample)
  hipError_t err;
  dim3 blocks(DIVUP(npoints * nsample, THREADS_PER_BLOCK), c,
              b);  // blockIdx.x(col), blockIdx.y(row)
  dim3 threads(THREADS_PER_BLOCK);

  group_points_kernel<<<blocks, threads, 0, stream>>>(b, c, n, npoints, nsample,
                                                      points, idx, out);
  // cudaDeviceSynchronize();  // for using printf in kernel function
  err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
    exit(-1);
  }
}
