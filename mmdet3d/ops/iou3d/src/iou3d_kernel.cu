// Modified from
// https://github.com/open-mmlab/OpenPCDet/blob/master/pcdet/ops/iou3d_nms/src/iou3d_nms_kernel.cu

/*
3D IoU Calculation and Rotated NMS(modified from 2D NMS written by others)
Written by Shaoshuai Shi
All Rights Reserved 2019-2020.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#define THREADS_PER_BLOCK 16
#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))

//#define DEBUG
const int THREADS_PER_BLOCK_NMS = sizeof(unsigned long long) * 8;
__device__ const float EPS = 1e-8;
struct Point {
  float x, y;
  __device__ Point() {}
  __device__ Point(double _x, double _y) { x = _x, y = _y; }

  __device__ void set(float _x, float _y) {
    x = _x;
    y = _y;
  }

  __device__ Point operator+(const Point &b) const {
    return Point(x + b.x, y + b.y);
  }

  __device__ Point operator-(const Point &b) const {
    return Point(x - b.x, y - b.y);
  }
};

__device__ inline float cross(const Point &a, const Point &b) {
  return a.x * b.y - a.y * b.x;
}

__device__ inline float cross(const Point &p1, const Point &p2,
                              const Point &p0) {
  return (p1.x - p0.x) * (p2.y - p0.y) - (p2.x - p0.x) * (p1.y - p0.y);
}

__device__ int check_rect_cross(const Point &p1, const Point &p2,
                                const Point &q1, const Point &q2) {
  int ret = min(p1.x, p2.x) <= max(q1.x, q2.x) &&
            min(q1.x, q2.x) <= max(p1.x, p2.x) &&
            min(p1.y, p2.y) <= max(q1.y, q2.y) &&
            min(q1.y, q2.y) <= max(p1.y, p2.y);
  return ret;
}

__device__ inline int check_in_box2d(const float *box, const Point &p) {
  // params: box (5) [x1, y1, x2, y2, angle]
  const float MARGIN = 1e-5;

  float center_x = (box[0] + box[2]) / 2;
  float center_y = (box[1] + box[3]) / 2;
  float angle_cos = cos(-box[4]),
        angle_sin =
            sin(-box[4]);  // rotate the point in the opposite direction of box
  float rot_x =
      (p.x - center_x) * angle_cos - (p.y - center_y) * angle_sin + center_x;
  float rot_y =
      (p.x - center_x) * angle_sin + (p.y - center_y) * angle_cos + center_y;
#ifdef DEBUG
  printf("box: (%.3f, %.3f, %.3f, %.3f, %.3f)\n", box[0], box[1], box[2],
         box[3], box[4]);
  printf(
      "center: (%.3f, %.3f), cossin(%.3f, %.3f), src(%.3f, %.3f), rot(%.3f, "
      "%.3f)\n",
      center_x, center_y, angle_cos, angle_sin, p.x, p.y, rot_x, rot_y);
#endif
  return (rot_x > box[0] - MARGIN && rot_x < box[2] + MARGIN &&
          rot_y > box[1] - MARGIN && rot_y < box[3] + MARGIN);
}

__device__ inline int intersection(const Point &p1, const Point &p0,
                                   const Point &q1, const Point &q0,
                                   Point &ans) {
  // fast exclusion
  if (check_rect_cross(p0, p1, q0, q1) == 0) return 0;

  // check cross standing
  float s1 = cross(q0, p1, p0);
  float s2 = cross(p1, q1, p0);
  float s3 = cross(p0, q1, q0);
  float s4 = cross(q1, p1, q0);

  if (!(s1 * s2 > 0 && s3 * s4 > 0)) return 0;

  // calculate intersection of two lines
  float s5 = cross(q1, p1, p0);
  if (fabs(s5 - s1) > EPS) {
    ans.x = (s5 * q0.x - s1 * q1.x) / (s5 - s1);
    ans.y = (s5 * q0.y - s1 * q1.y) / (s5 - s1);

  } else {
    float a0 = p0.y - p1.y, b0 = p1.x - p0.x, c0 = p0.x * p1.y - p1.x * p0.y;
    float a1 = q0.y - q1.y, b1 = q1.x - q0.x, c1 = q0.x * q1.y - q1.x * q0.y;
    float D = a0 * b1 - a1 * b0;

    ans.x = (b0 * c1 - b1 * c0) / D;
    ans.y = (a1 * c0 - a0 * c1) / D;
  }

  return 1;
}

__device__ inline void rotate_around_center(const Point &center,
                                            const float angle_cos,
                                            const float angle_sin, Point &p) {
  float new_x =
      (p.x - center.x) * angle_cos - (p.y - center.y) * angle_sin + center.x;
  float new_y =
      (p.x - center.x) * angle_sin + (p.y - center.y) * angle_cos + center.y;
  p.set(new_x, new_y);
}

__device__ inline int point_cmp(const Point &a, const Point &b,
                                const Point &center) {
  return atan2(a.y - center.y, a.x - center.x) >
         atan2(b.y - center.y, b.x - center.x);
}

__device__ inline float box_overlap(const float *box_a, const float *box_b) {
  // params: box_a (5) [x1, y1, x2, y2, angle]
  // params: box_b (5) [x1, y1, x2, y2, angle]

  float a_x1 = box_a[0], a_y1 = box_a[1], a_x2 = box_a[2], a_y2 = box_a[3],
        a_angle = box_a[4];
  float b_x1 = box_b[0], b_y1 = box_b[1], b_x2 = box_b[2], b_y2 = box_b[3],
        b_angle = box_b[4];

  Point center_a((a_x1 + a_x2) / 2, (a_y1 + a_y2) / 2);
  Point center_b((b_x1 + b_x2) / 2, (b_y1 + b_y2) / 2);
#ifdef DEBUG
  printf(
      "a: (%.3f, %.3f, %.3f, %.3f, %.3f), b: (%.3f, %.3f, %.3f, %.3f, %.3f)\n",
      a_x1, a_y1, a_x2, a_y2, a_angle, b_x1, b_y1, b_x2, b_y2, b_angle);
  printf("center a: (%.3f, %.3f), b: (%.3f, %.3f)\n", center_a.x, center_a.y,
         center_b.x, center_b.y);
#endif

  Point box_a_corners[5];
  box_a_corners[0].set(a_x1, a_y1);
  box_a_corners[1].set(a_x2, a_y1);
  box_a_corners[2].set(a_x2, a_y2);
  box_a_corners[3].set(a_x1, a_y2);

  Point box_b_corners[5];
  box_b_corners[0].set(b_x1, b_y1);
  box_b_corners[1].set(b_x2, b_y1);
  box_b_corners[2].set(b_x2, b_y2);
  box_b_corners[3].set(b_x1, b_y2);

  // get oriented corners
  float a_angle_cos = cos(a_angle), a_angle_sin = sin(a_angle);
  float b_angle_cos = cos(b_angle), b_angle_sin = sin(b_angle);

  for (int k = 0; k < 4; k++) {
#ifdef DEBUG
    printf("before corner %d: a(%.3f, %.3f), b(%.3f, %.3f) \n", k,
           box_a_corners[k].x, box_a_corners[k].y, box_b_corners[k].x,
           box_b_corners[k].y);
#endif
    rotate_around_center(center_a, a_angle_cos, a_angle_sin, box_a_corners[k]);
    rotate_around_center(center_b, b_angle_cos, b_angle_sin, box_b_corners[k]);
#ifdef DEBUG
    printf("corner %d: a(%.3f, %.3f), b(%.3f, %.3f) \n", k, box_a_corners[k].x,
           box_a_corners[k].y, box_b_corners[k].x, box_b_corners[k].y);
#endif
  }

  box_a_corners[4] = box_a_corners[0];
  box_b_corners[4] = box_b_corners[0];

  // get intersection of lines
  Point cross_points[16];
  Point poly_center;
  int cnt = 0, flag = 0;

  poly_center.set(0, 0);
  for (int i = 0; i < 4; i++) {
    for (int j = 0; j < 4; j++) {
      flag = intersection(box_a_corners[i + 1], box_a_corners[i],
                          box_b_corners[j + 1], box_b_corners[j],
                          cross_points[cnt]);
      if (flag) {
        poly_center = poly_center + cross_points[cnt];
        cnt++;
      }
    }
  }

  // check corners
  for (int k = 0; k < 4; k++) {
    if (check_in_box2d(box_a, box_b_corners[k])) {
      poly_center = poly_center + box_b_corners[k];
      cross_points[cnt] = box_b_corners[k];
      cnt++;
    }
    if (check_in_box2d(box_b, box_a_corners[k])) {
      poly_center = poly_center + box_a_corners[k];
      cross_points[cnt] = box_a_corners[k];
      cnt++;
    }
  }

  poly_center.x /= cnt;
  poly_center.y /= cnt;

  // sort the points of polygon
  Point temp;
  for (int j = 0; j < cnt - 1; j++) {
    for (int i = 0; i < cnt - j - 1; i++) {
      if (point_cmp(cross_points[i], cross_points[i + 1], poly_center)) {
        temp = cross_points[i];
        cross_points[i] = cross_points[i + 1];
        cross_points[i + 1] = temp;
      }
    }
  }

#ifdef DEBUG
  printf("cnt=%d\n", cnt);
  for (int i = 0; i < cnt; i++) {
    printf("All cross point %d: (%.3f, %.3f)\n", i, cross_points[i].x,
           cross_points[i].y);
  }
#endif

  // get the overlap areas
  float area = 0;
  for (int k = 0; k < cnt - 1; k++) {
    area += cross(cross_points[k] - cross_points[0],
                  cross_points[k + 1] - cross_points[0]);
  }

  return fabs(area) / 2.0;
}

__device__ inline float iou_bev(const float *box_a, const float *box_b) {
  // params: box_a (5) [x1, y1, x2, y2, angle]
  // params: box_b (5) [x1, y1, x2, y2, angle]
  float sa = (box_a[2] - box_a[0]) * (box_a[3] - box_a[1]);
  float sb = (box_b[2] - box_b[0]) * (box_b[3] - box_b[1]);
  float s_overlap = box_overlap(box_a, box_b);
  return s_overlap / fmaxf(sa + sb - s_overlap, EPS);
}

__global__ void boxes_overlap_kernel(const int num_a, const float *boxes_a,
                                     const int num_b, const float *boxes_b,
                                     float *ans_overlap) {
  const int a_idx = blockIdx.y * THREADS_PER_BLOCK + threadIdx.y;
  const int b_idx = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;

  if (a_idx >= num_a || b_idx >= num_b) {
    return;
  }
  const float *cur_box_a = boxes_a + a_idx * 5;
  const float *cur_box_b = boxes_b + b_idx * 5;
  float s_overlap = box_overlap(cur_box_a, cur_box_b);
  ans_overlap[a_idx * num_b + b_idx] = s_overlap;
}

__global__ void boxes_iou_bev_kernel(const int num_a, const float *boxes_a,
                                     const int num_b, const float *boxes_b,
                                     float *ans_iou) {
  const int a_idx = blockIdx.y * THREADS_PER_BLOCK + threadIdx.y;
  const int b_idx = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;

  if (a_idx >= num_a || b_idx >= num_b) {
    return;
  }

  const float *cur_box_a = boxes_a + a_idx * 5;
  const float *cur_box_b = boxes_b + b_idx * 5;
  float cur_iou_bev = iou_bev(cur_box_a, cur_box_b);
  ans_iou[a_idx * num_b + b_idx] = cur_iou_bev;
}

__global__ void nms_kernel(const int boxes_num, const float nms_overlap_thresh,
                           const float *boxes, unsigned long long *mask) {
  // params: boxes (N, 5) [x1, y1, x2, y2, ry]
  // params: mask (N, N/THREADS_PER_BLOCK_NMS)

  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  const int row_size = fminf(boxes_num - row_start * THREADS_PER_BLOCK_NMS,
                             THREADS_PER_BLOCK_NMS);
  const int col_size = fminf(boxes_num - col_start * THREADS_PER_BLOCK_NMS,
                             THREADS_PER_BLOCK_NMS);

  __shared__ float block_boxes[THREADS_PER_BLOCK_NMS * 5];

  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 5 + 0] =
        boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 5 + 0];
    block_boxes[threadIdx.x * 5 + 1] =
        boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 5 + 1];
    block_boxes[threadIdx.x * 5 + 2] =
        boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 5 + 2];
    block_boxes[threadIdx.x * 5 + 3] =
        boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 5 + 3];
    block_boxes[threadIdx.x * 5 + 4] =
        boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 5 + 4];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = THREADS_PER_BLOCK_NMS * row_start + threadIdx.x;
    const float *cur_box = boxes + cur_box_idx * 5;

    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (iou_bev(cur_box, block_boxes + i * 5) > nms_overlap_thresh) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = DIVUP(boxes_num, THREADS_PER_BLOCK_NMS);
    mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

__device__ inline float iou_normal(float const *const a, float const *const b) {
  float left = fmaxf(a[0], b[0]), right = fminf(a[2], b[2]);
  float top = fmaxf(a[1], b[1]), bottom = fminf(a[3], b[3]);
  float width = fmaxf(right - left, 0.f), height = fmaxf(bottom - top, 0.f);
  float interS = width * height;
  float Sa = (a[2] - a[0]) * (a[3] - a[1]);
  float Sb = (b[2] - b[0]) * (b[3] - b[1]);
  return interS / fmaxf(Sa + Sb - interS, EPS);
}

__global__ void nms_normal_kernel(const int boxes_num,
                                  const float nms_overlap_thresh,
                                  const float *boxes,
                                  unsigned long long *mask) {
  // params: boxes (N, 5) [x1, y1, x2, y2, ry]
  // params: mask (N, N/THREADS_PER_BLOCK_NMS)

  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  const int row_size = fminf(boxes_num - row_start * THREADS_PER_BLOCK_NMS,
                             THREADS_PER_BLOCK_NMS);
  const int col_size = fminf(boxes_num - col_start * THREADS_PER_BLOCK_NMS,
                             THREADS_PER_BLOCK_NMS);

  __shared__ float block_boxes[THREADS_PER_BLOCK_NMS * 5];

  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 5 + 0] =
        boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 5 + 0];
    block_boxes[threadIdx.x * 5 + 1] =
        boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 5 + 1];
    block_boxes[threadIdx.x * 5 + 2] =
        boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 5 + 2];
    block_boxes[threadIdx.x * 5 + 3] =
        boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 5 + 3];
    block_boxes[threadIdx.x * 5 + 4] =
        boxes[(THREADS_PER_BLOCK_NMS * col_start + threadIdx.x) * 5 + 4];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = THREADS_PER_BLOCK_NMS * row_start + threadIdx.x;
    const float *cur_box = boxes + cur_box_idx * 5;

    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (iou_normal(cur_box, block_boxes + i * 5) > nms_overlap_thresh) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = DIVUP(boxes_num, THREADS_PER_BLOCK_NMS);
    mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

void boxesoverlapLauncher(const int num_a, const float *boxes_a,
                          const int num_b, const float *boxes_b,
                          float *ans_overlap) {
  dim3 blocks(
      DIVUP(num_b, THREADS_PER_BLOCK),
      DIVUP(num_a, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
  dim3 threads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);

  boxes_overlap_kernel<<<blocks, threads>>>(num_a, boxes_a, num_b, boxes_b,
                                            ans_overlap);
#ifdef DEBUG
  cudaDeviceSynchronize();  // for using printf in kernel function
#endif
}

void boxesioubevLauncher(const int num_a, const float *boxes_a, const int num_b,
                         const float *boxes_b, float *ans_iou) {
  dim3 blocks(
      DIVUP(num_b, THREADS_PER_BLOCK),
      DIVUP(num_a, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
  dim3 threads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);

  boxes_iou_bev_kernel<<<blocks, threads>>>(num_a, boxes_a, num_b, boxes_b,
                                            ans_iou);
}

void nmsLauncher(const float *boxes, unsigned long long *mask, int boxes_num,
                 float nms_overlap_thresh) {
  dim3 blocks(DIVUP(boxes_num, THREADS_PER_BLOCK_NMS),
              DIVUP(boxes_num, THREADS_PER_BLOCK_NMS));
  dim3 threads(THREADS_PER_BLOCK_NMS);
  nms_kernel<<<blocks, threads>>>(boxes_num, nms_overlap_thresh, boxes, mask);
}

void nmsNormalLauncher(const float *boxes, unsigned long long *mask,
                       int boxes_num, float nms_overlap_thresh) {
  dim3 blocks(DIVUP(boxes_num, THREADS_PER_BLOCK_NMS),
              DIVUP(boxes_num, THREADS_PER_BLOCK_NMS));
  dim3 threads(THREADS_PER_BLOCK_NMS);
  nms_normal_kernel<<<blocks, threads>>>(boxes_num, nms_overlap_thresh, boxes,
                                         mask);
}
