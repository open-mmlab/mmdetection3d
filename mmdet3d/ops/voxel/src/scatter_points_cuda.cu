#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/types.h>

#include <ATen/cuda/CUDAApplyUtils.cuh>

typedef enum { SUM = 0, MEAN = 1, MAX = 2 } reduce_t;

#define CHECK_CUDA(x) \
  TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) \
  TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) \
  CHECK_CUDA(x);       \
  CHECK_CONTIGUOUS(x)

namespace {
int const threadsPerBlock = 512;
int const maxGridDim = 50000;
}  // namespace

__device__ __forceinline__ static void reduceMax(float *address, float val) {
  int *address_as_i = reinterpret_cast<int *>(address);
  int old = *address_as_i, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_i, assumed,
                    __float_as_int(fmaxf(val, __int_as_float(assumed))));
  } while (assumed != old || __int_as_float(old) < val);
}

__device__ __forceinline__ static void reduceMax(double *address, double val) {
  unsigned long long *address_as_ull =
      reinterpret_cast<unsigned long long *>(address);
  unsigned long long old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(
        address_as_ull, assumed,
        __double_as_longlong(fmax(val, __longlong_as_double(assumed))));
  } while (assumed != old || __longlong_as_double(old) < val);
}

// get rid of meaningless warnings when compiling host code
#ifdef __CUDA_ARCH__
__device__ __forceinline__ static void reduceAdd(float *address, float val) {
#if (__CUDA_ARCH__ < 200)
#warning \
    "compute capability lower than 2.x. fall back to use CAS version of atomicAdd for float32"
  int *address_as_i = reinterpret_cast<int *>(address);
  int old = *address_as_i, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_i, assumed,
                    __float_as_int(val + __int_as_float(assumed)));
  } while (assumed != old);
#else
  atomicAdd(address, val);
#endif
}

__device__ __forceinline__ static void reduceAdd(double *address, double val) {
#if (__CUDA_ARCH__ < 600)
#warning \
    "compute capability lower than 6.x. fall back to use CAS version of atomicAdd for float64"
  unsigned long long *address_as_ull =
      reinterpret_cast<unsigned long long *>(address);
  unsigned long long old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
#else
  atomicAdd(address, val);
#endif
}
#endif

template <typename T_int>
__global__ void coors_id_kernel(const T_int *coors, const T_int *dim,
                                int64_t *coors_id, const int num_input,
                                const int NDim) {
  for (int x = blockIdx.x * blockDim.x + threadIdx.x; x < num_input;
       x += gridDim.x * blockDim.x) {
    const T_int *coor_x = coors + x * NDim;
    auto coor_id = 0;
    for (int i = 0; i < NDim && coor_id != -1; i++) {
      coor_id *= dim[i];
      auto t = static_cast<int64_t>(coor_x[i]);
      coor_id = (t < 0) ? -1 : coor_id + t;
    }
    coors_id[x] = coor_id;
  }
}

template <typename T_int>
__global__ void coors_map_init_kernel(const int64_t *coors_id,
                                      const T_int *coors_id_argsort,
                                      int32_t *coors_map, const int num_input) {
  for (int x = blockIdx.x * blockDim.x + threadIdx.x; x < num_input;
       x += gridDim.x * blockDim.x) {
    auto here = coors_id[coors_id_argsort[x]];
    if (x == 0) {
      if (here == -1) {  // there is invalid points
        coors_map[0] = -1;
      } else {
        coors_map[0] = 0;
      }
      continue;
    }
    auto left = coors_id[coors_id_argsort[x - 1]];
    coors_map[x] = (left < here) ? 1 : 0;
  }
}

template <typename T, typename T_int>
__global__ void feats_reduce_kernel(
    const T *feats, const T_int *coors, int32_t *coors_map,
    int32_t *reduce_count,  // shall be 0 at initialization
    T *reduced_feats,       // shall be 0 at initialization
    T_int *out_coors, const int num_input, const int num_feats, const int NDim,
    const reduce_t reduce_type) {
  for (int x = blockIdx.x * blockDim.x + threadIdx.x; x < num_input;
       x += gridDim.x * blockDim.x) {
    int32_t reduce_to = coors_map[x];
    if (reduce_to == -1) continue;

    const T_int *coors_offset = coors + x * NDim;
    T_int *out_coors_offset = out_coors + reduce_to * NDim;
    for (int i = 0; i < NDim; i++) {
      out_coors_offset[i] = coors_offset[i];
    }

    const T *feats_offset = feats + x * num_feats;
    T *reduced_feats_offset = reduced_feats + reduce_to * num_feats;
    if (reduce_type == reduce_t::MAX) {
      for (int i = 0; i < num_feats; i++) {
        reduceMax(&reduced_feats_offset[i], feats_offset[i]);
      }
    } else {
      if (reduce_type == reduce_t::MEAN) {
        atomicAdd(&reduce_count[reduce_to], static_cast<int32_t>(1));
      }
      for (int i = 0; i < num_feats; i++) {
        reduceAdd(&reduced_feats_offset[i], feats_offset[i]);
      }
    }
  }
}

template <typename T>
__global__ void add_reduce_traceback_grad_kernel(
    T *grad_feats, const T *grad_reduced_feats, const int32_t *coors_map,
    const int32_t *reduce_count, const int num_input, const int num_feats,
    const reduce_t reduce_type) {
  for (int x = blockIdx.x * blockDim.x + threadIdx.x; x < num_input;
       x += gridDim.x * blockDim.x) {
    int32_t reduce_to = coors_map[x];
    if (reduce_to == -1) {
      continue;
    }

    const int input_offset = x * num_feats;
    T *grad_feats_offset = grad_feats + input_offset;
    const int reduced_offset = reduce_to * num_feats;
    const T *grad_reduced_feats_offset = grad_reduced_feats + reduced_offset;

    if (reduce_type == reduce_t::SUM) {
      for (int i = 0; i < num_feats; i++) {
        grad_feats_offset[i] = grad_reduced_feats_offset[i];
      }
    } else if (reduce_type == reduce_t::MEAN) {
      for (int i = 0; i < num_feats; i++) {
        grad_feats_offset[i] = grad_reduced_feats_offset[i] /
                               static_cast<T>(reduce_count[reduce_to]);
      }
    }
  }
}

template <typename T>
__global__ void max_reduce_traceback_scatter_idx_kernel(
    const T *feats, const T *reduced_feats, int32_t *reduce_from,
    const int32_t *coors_map, const int num_input, const int num_feats) {
  for (int x = blockIdx.x * blockDim.x + threadIdx.x; x < num_input;
       x += gridDim.x * blockDim.x) {
    int32_t reduce_to = coors_map[x];

    const int input_offset = x * num_feats;
    const T *feats_offset = feats + input_offset;

    if (reduce_to == -1) {
      continue;
    }

    const int reduced_offset = reduce_to * num_feats;
    const T *reduced_feats_offset = reduced_feats + reduced_offset;
    int32_t *reduce_from_offset = reduce_from + reduced_offset;

    for (int i = 0; i < num_feats; i++) {
      if (feats_offset[i] == reduced_feats_offset[i]) {
        atomicMin(&reduce_from_offset[i], static_cast<int32_t>(x));
      }
    }
  }
}

template <typename T>
__global__ void max_reduce_scatter_grad_kernel(T *grad_feats,
                                               const T *grad_reduced_feats,
                                               const int32_t *reduce_from,
                                               const int num_reduced,
                                               const int num_feats) {
  for (int x = blockIdx.x * blockDim.x + threadIdx.x; x < num_reduced;
       x += gridDim.x * blockDim.x) {
    const int reduced_offset = x * num_feats;
    const int32_t *scatter_to_offset = reduce_from + reduced_offset;
    const T *grad_reduced_feats_offset = grad_reduced_feats + reduced_offset;

    for (int i = 0; i < num_feats; i++) {
      grad_feats[scatter_to_offset[i] * num_feats + i] =
          grad_reduced_feats_offset[i];
    }
  }
}

namespace voxelization {

std::vector<at::Tensor> dynamic_point_to_voxel_forward_gpu(
    const at::Tensor &feats, const at::Tensor &coors,
    const reduce_t reduce_type) {
  CHECK_INPUT(feats);
  CHECK_INPUT(coors);

  const int NDim = coors.size(1);
  const int num_input = feats.size(0);
  const int num_feats = feats.size(1);

  auto coors_id = at::empty({num_input}, coors.options().dtype(torch::kInt64));
  auto coor_space_dim = std::get<0>(coors.max(0)) + 1;
  auto coors_map_sorted =
      at::empty({num_input}, coors.options().dtype(torch::kInt32));
  auto coors_map = at::empty({num_input}, coors.options().dtype(torch::kInt32));
  auto num_coors = at::zeros({1}, coors.options().dtype(torch::kInt32));

  AT_DISPATCH_INTEGRAL_TYPES(
      coors.scalar_type(), "coors_id_kernel", ([&] {
        dim3 blocks(std::min(at::cuda::ATenCeilDiv(num_input, threadsPerBlock),
                             maxGridDim));
        dim3 threads(threadsPerBlock);
        coors_id_kernel<<<blocks, threads>>>(
            coors.data_ptr<scalar_t>(), coor_space_dim.data_ptr<scalar_t>(),
            coors_id.data_ptr<int64_t>(), num_input, NDim);
      }));
  AT_CUDA_CHECK(hipGetLastError());

  auto coors_id_argsort = coors_id.argsort();

  AT_DISPATCH_INTEGRAL_TYPES(
      coors_id_argsort.scalar_type(), "coors_map_init_kernel", ([&] {
        dim3 blocks(std::min(at::cuda::ATenCeilDiv(num_input, threadsPerBlock),
                             maxGridDim));
        dim3 threads(threadsPerBlock);
        coors_map_init_kernel<<<blocks, threads>>>(
            coors_id.data_ptr<int64_t>(), coors_id_argsort.data_ptr<scalar_t>(),
            coors_map_sorted.data_ptr<int32_t>(), num_input);
      }));
  AT_CUDA_CHECK(hipGetLastError());

  coors_map_sorted = coors_map_sorted.cumsum(0, torch::kInt32);
  coors_map.index_put_({coors_id_argsort}, coors_map_sorted);

  const int num_coors_cpu =
      coors_map_sorted[-1].cpu().data_ptr<int32_t>()[0] + 1;
  auto out_coors = at::empty({num_coors_cpu, NDim}, coors.options());
  auto reduced_feats = at::empty({num_coors_cpu, num_feats}, feats.options());
  auto reduce_count =
      at::zeros({num_coors_cpu}, coors.options().dtype(torch::kInt32));

  AT_DISPATCH_FLOATING_TYPES(
      feats.scalar_type(), "feats_reduce_kernel", ([&] {
        using F_t = scalar_t;
        AT_DISPATCH_INTEGRAL_TYPES(
            coors.scalar_type(), "feats_reduce_kernel", ([&] {
              using I_t = scalar_t;

              if (reduce_type == reduce_t::MAX)
                reduced_feats.fill_(-std::numeric_limits<F_t>::infinity());
              else
                reduced_feats.fill_(static_cast<F_t>(0));

              dim3 blocks(
                  std::min(at::cuda::ATenCeilDiv(num_input, threadsPerBlock),
                           maxGridDim));
              dim3 threads(threadsPerBlock);
              feats_reduce_kernel<<<blocks, threads>>>(
                  feats.data_ptr<F_t>(), coors.data_ptr<I_t>(),
                  coors_map.data_ptr<int32_t>(),
                  reduce_count.data_ptr<int32_t>(),
                  reduced_feats.data_ptr<F_t>(), out_coors.data_ptr<I_t>(),
                  num_input, num_feats, NDim, reduce_type);
              if (reduce_type == reduce_t::MEAN)
                reduced_feats /=
                    reduce_count.unsqueeze(-1).to(reduced_feats.dtype());
            }));
      }));
  AT_CUDA_CHECK(hipGetLastError());

  return {reduced_feats, out_coors, coors_map, reduce_count};
}

void dynamic_point_to_voxel_backward_gpu(at::Tensor &grad_feats,
                                         const at::Tensor &grad_reduced_feats,
                                         const at::Tensor &feats,
                                         const at::Tensor &reduced_feats,
                                         const at::Tensor &coors_map,
                                         const at::Tensor &reduce_count,
                                         const reduce_t reduce_type) {
  CHECK_INPUT(grad_feats);
  CHECK_INPUT(grad_reduced_feats);
  CHECK_INPUT(feats);
  CHECK_INPUT(reduced_feats);
  CHECK_INPUT(coors_map);
  CHECK_INPUT(reduce_count);

  const int num_input = feats.size(0);
  const int num_reduced = reduced_feats.size(0);
  const int num_feats = feats.size(1);

  grad_feats.fill_(0);
  // copy voxel grad to points

  if (reduce_type == reduce_t::MEAN || reduce_type == reduce_t::SUM) {
    AT_DISPATCH_FLOATING_TYPES(
        grad_reduced_feats.scalar_type(), "add_reduce_traceback_grad_kernel",
        ([&] {
          dim3 blocks(std::min(
              at::cuda::ATenCeilDiv(num_input, threadsPerBlock), maxGridDim));
          dim3 threads(threadsPerBlock);
          add_reduce_traceback_grad_kernel<<<blocks, threads>>>(
              grad_feats.data_ptr<scalar_t>(),
              grad_reduced_feats.data_ptr<scalar_t>(),
              coors_map.data_ptr<int32_t>(), reduce_count.data_ptr<int32_t>(),
              num_input, num_feats, reduce_type);
        }));
    AT_CUDA_CHECK(hipGetLastError());
  } else {
    auto reduce_from = at::full({num_reduced, num_feats}, num_input,
                                coors_map.options().dtype(torch::kInt32));
    AT_DISPATCH_FLOATING_TYPES(
        grad_reduced_feats.scalar_type(),
        "max_reduce_traceback_scatter_idx_kernel", ([&] {
          dim3 blocks(std::min(
              at::cuda::ATenCeilDiv(num_input, threadsPerBlock), maxGridDim));
          dim3 threads(threadsPerBlock);
          max_reduce_traceback_scatter_idx_kernel<<<blocks, threads>>>(
              feats.data_ptr<scalar_t>(), reduced_feats.data_ptr<scalar_t>(),
              reduce_from.data_ptr<int32_t>(), coors_map.data_ptr<int32_t>(),
              num_input, num_feats);
        }));
    AT_CUDA_CHECK(hipGetLastError());

    AT_DISPATCH_FLOATING_TYPES(
        grad_reduced_feats.scalar_type(),
        "max_reduce_traceback_scatter_idx_kernel", ([&] {
          dim3 blocks(std::min(
              at::cuda::ATenCeilDiv(num_reduced, threadsPerBlock), maxGridDim));
          dim3 threads(threadsPerBlock);
          max_reduce_scatter_grad_kernel<<<blocks, threads>>>(
              grad_feats.data_ptr<scalar_t>(),
              grad_reduced_feats.data_ptr<scalar_t>(),
              reduce_from.data_ptr<int32_t>(), num_reduced, num_feats);
        }));
    AT_CUDA_CHECK(hipGetLastError());
  }
  return;
}

}  // namespace voxelization
