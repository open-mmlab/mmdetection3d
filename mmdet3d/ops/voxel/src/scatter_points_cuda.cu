#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <torch/types.h>

#include <ATen/cuda/CUDAApplyUtils.cuh>

#define CHECK_CUDA(x) \
  TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) \
  TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) \
  CHECK_CUDA(x);       \
  CHECK_CONTIGUOUS(x)

namespace {
int const threadsPerBlock = sizeof(unsigned long long) * 8;
}

template <typename T, typename T_int>
__global__ void scatter_point_to_voxel_kernel(
    const T* points, T_int* coor, T_int* point_to_voxelidx,
    T_int* coor_to_voxelidx, T* voxels, T_int* coors, const int num_features,
    const int num_points, const int max_points, const int NDim) {
  const int index = blockIdx.x * threadsPerBlock + threadIdx.x;
  if (index >= num_points) return;

  int num = point_to_voxelidx[index];
  int voxelidx = coor_to_voxelidx[index];
  if (num > -1 && voxelidx > -1) {
    const int feature_per_thread = 1;

    int start = threadIdx.y * feature_per_thread;
    auto voxels_offset =
        voxels + voxelidx * max_points * num_features + num * num_features;
    auto points_offset = points + index * num_features;
    for (int k = start; k < start + feature_per_thread; k++) {
      voxels_offset[k] = points_offset[k];
    }
    if (num == 0 && start < NDim) {
      auto coors_offset = coors + voxelidx * NDim;
      auto coor_offset = coor + index * NDim;
      for (int k = start; k < NDim; k++) {
        coors_offset[k] = coor_offset[k];
      }
    }
  }
}

template <typename T, typename T_int>
__global__ void map_voxel_to_point_kernel(
    T* points, T* voxels, T_int* point_to_voxelidx, T_int* coor_to_voxelidx,
    const int num_features, const int num_points, const int max_points) {
  const int index = blockIdx.x * threadsPerBlock + threadIdx.x;
  if (index >= num_points) return;
  auto num = point_to_voxelidx[index];
  if (num > -1) {
    const int feature_per_thread = 1;
    auto voxelidx = coor_to_voxelidx[index];

    int start = threadIdx.y * feature_per_thread;
    auto voxels_offset =
        voxels + voxelidx * max_points * num_features + num * num_features;
    auto points_offset = points + index * num_features;
    for (int k = start; k < start + feature_per_thread; k++) {
      points_offset[k] = voxels_offset[k];
    }
  }
}

template <typename T_int>
__global__ void point_to_voxelidx_kernel(const T_int* coor,
                                         T_int* point_to_voxelidx,
                                         T_int* point_to_pointidx,
                                         const int num_points, const int NDim) {
  const int index = blockIdx.x * threadsPerBlock + threadIdx.x;
  auto coor_offset = coor + index * NDim;
  // skip invalid points
  if ((index >= num_points) || (coor_offset[0] == -1)) return;

  int num = 0;
  int coor_x = coor_offset[0];
  int coor_y = coor_offset[1];
  int coor_z = coor_offset[2];
  // only calculate the coors before this coor[index]
  for (int i = 0; i < index; ++i) {
    auto prev_coor = coor + i * NDim;
    if (prev_coor[0] == -1) continue;

    // record voxel
    if ((prev_coor[0] == coor_x) && (prev_coor[1] == coor_y) &&
        (prev_coor[2] == coor_z)) {
      num++;
      if (num == 1) {
        point_to_pointidx[index] = i;
      }
    }
  }
  if (num == 0) {
    point_to_pointidx[index] = index;
  }
  point_to_voxelidx[index] = num;
}

template <typename T_int>
__global__ void determin_voxel_num(
    const T_int* coor, T_int* num_points_per_voxel, T_int* point_to_voxelidx,
    T_int* point_to_pointidx, T_int* coor_to_voxelidx, T_int* voxel_num,
    T_int* max_points, const int num_points, const int NDim) {
  // only calculate the coors before this coor[index]
  for (int i = 0; i < num_points; ++i) {
    auto coor_offset = coor + i * NDim;
    if (coor_offset[0] == -1) continue;
    int point_pos_in_voxel = point_to_voxelidx[i];
    // record voxel
    if (point_pos_in_voxel == -1) {
      // out of max_points or invalid point
      printf("point_pos_in_voxel == -1, point:%d", i);
      continue;
    } else if (point_pos_in_voxel == 0) {
      // record new voxel
      int voxelidx = voxel_num[0];
      voxel_num[0] += 1;
      coor_to_voxelidx[i] = voxelidx;
      num_points_per_voxel[voxelidx] = 1;
    } else {
      int point_idx = point_to_pointidx[i];
      int voxelidx = coor_to_voxelidx[point_idx];
      if (voxelidx != -1) {
        num_points_per_voxel[voxelidx] += 1;
        coor_to_voxelidx[i] = voxelidx;
        max_points[0] = max(max_points[0], point_pos_in_voxel + 1);
      } else {
        printf("voxelidx = -1, point:%d", i);
      }
    }
  }
}

namespace voxelization {

std::vector<at::Tensor> dynamic_point_to_voxel_forward_gpu(
    const at::Tensor& points, const at::Tensor& voxel_mapping,
    const std::vector<float> voxel_size, const std::vector<float> coors_range) {
  CHECK_INPUT(points);
  at::cuda::CUDAGuard device_guard(points.device());

  const int NDim = voxel_mapping.size(1);
  const int num_points = points.size(0);
  const int num_features = points.size(1);

  std::vector<int> grid_size(NDim);
  for (int i = 0; i < NDim; ++i) {
    grid_size[i] =
        round((coors_range[NDim + i] - coors_range[i]) / voxel_size[i]);
  }

  // assume the mapping is already given
  auto point_to_pointidx = -at::ones(
      {
          num_points,
      },
      voxel_mapping.options());
  auto point_to_voxelidx = -at::ones(
      {
          num_points,
      },
      voxel_mapping.options());
  auto max_points = at::zeros(
      {
          1,
      },
      voxel_mapping.options());  // must be zero from the begining

  int col_blocks = at::cuda::ATenCeilDiv(num_points, threadsPerBlock);
  dim3 blocks(col_blocks);
  dim3 threads(threadsPerBlock);
  hipStream_t map_stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES(
      voxel_mapping.scalar_type(), "determin_duplicate", ([&] {
        point_to_voxelidx_kernel<int><<<blocks, threads, 0, map_stream>>>(
            voxel_mapping.data_ptr<int>(), point_to_voxelidx.data_ptr<int>(),
            point_to_pointidx.data_ptr<int>(), num_points, NDim);
      }));
  hipDeviceSynchronize();
  AT_CUDA_CHECK(hipGetLastError());

  // make the logic in the CUDA device could accelerate about 10 times
  auto num_points_per_voxel = at::zeros(
      {
          num_points,
      },
      voxel_mapping.options());
  auto coor_to_voxelidx = -at::ones(
      {
          num_points,
      },
      voxel_mapping.options());
  auto voxel_num = at::zeros(
      {
          1,
      },
      voxel_mapping.options());  // must be zero from the begining
  hipStream_t logic_stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES(
      voxel_mapping.scalar_type(), "determin_duplicate", ([&] {
        determin_voxel_num<int><<<1, 1, 0, logic_stream>>>(
            voxel_mapping.data_ptr<int>(), num_points_per_voxel.data_ptr<int>(),
            point_to_voxelidx.data_ptr<int>(),
            point_to_pointidx.data_ptr<int>(), coor_to_voxelidx.data_ptr<int>(),
            voxel_num.data_ptr<int>(), max_points.data_ptr<int>(), num_points,
            NDim);
      }));
  hipDeviceSynchronize();
  AT_CUDA_CHECK(hipGetLastError());

  // some temporary data
  auto max_points_cpu = max_points.to(at::kCPU);
  int max_points_int = max_points_cpu.data_ptr<int>()[0];
  auto voxel_num_cpu = voxel_num.to(at::kCPU);
  int voxel_num_int = voxel_num_cpu.data_ptr<int>()[0];
  at::Tensor coors =
      at::zeros({voxel_num_int, NDim}, points.options().dtype(at::kInt));
  at::Tensor voxels = at::zeros({voxel_num_int, max_points_int, num_features},
                                points.options());

  // copy point features to voxels
  dim3 cp_threads(threadsPerBlock, num_features);
  hipStream_t cp_stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES(
      points.scalar_type(), "scatter_point_to_voxel", ([&] {
        scatter_point_to_voxel_kernel<float, int>
            <<<blocks, cp_threads, 0, cp_stream>>>(
                points.data_ptr<float>(), voxel_mapping.data_ptr<int>(),
                point_to_voxelidx.data_ptr<int>(),
                coor_to_voxelidx.data_ptr<int>(), voxels.data_ptr<float>(),
                coors.data_ptr<int>(), num_features, num_points, max_points_int,
                NDim);
      }));
  hipDeviceSynchronize();
  AT_CUDA_CHECK(hipGetLastError());

  at::Tensor num_points_per_voxel_out =
      num_points_per_voxel.slice(/*dim=*/0, /*start=*/0, /*end=*/voxel_num_int);
  return {voxels, coors, num_points_per_voxel_out, point_to_voxelidx,
          coor_to_voxelidx};
}

void dynamic_point_to_voxel_backward_gpu(at::Tensor& grad_input_points,
                                         const at::Tensor& grad_output_voxels,
                                         const at::Tensor& point_to_voxelidx,
                                         const at::Tensor& coor_to_voxelidx) {
  CHECK_INPUT(grad_input_points);
  CHECK_INPUT(grad_output_voxels);
  CHECK_INPUT(point_to_voxelidx);
  CHECK_INPUT(coor_to_voxelidx);
  at::cuda::CUDAGuard device_guard(grad_input_points.device());

  const int num_points = grad_input_points.size(0);
  const int num_features = grad_input_points.size(1);
  const int max_points = grad_output_voxels.size(1);

  // copy voxel grad to points
  int col_blocks = at::cuda::ATenCeilDiv(num_points, threadsPerBlock);
  dim3 blocks(col_blocks);
  dim3 cp_threads(threadsPerBlock, num_features);
  hipStream_t cp_stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES(grad_input_points.scalar_type(),
                        "scatter_point_to_voxel", ([&] {
                          map_voxel_to_point_kernel<float, int>
                              <<<blocks, cp_threads, 0, cp_stream>>>(
                                  grad_input_points.data_ptr<float>(),
                                  grad_output_voxels.data_ptr<float>(),
                                  point_to_voxelidx.data_ptr<int>(),
                                  coor_to_voxelidx.data_ptr<int>(),
                                  num_features, num_points, max_points);
                        }));
  hipDeviceSynchronize();
  AT_CUDA_CHECK(hipGetLastError());

  return;
}

}  // namespace voxelization
