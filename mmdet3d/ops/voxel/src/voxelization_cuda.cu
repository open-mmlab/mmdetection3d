#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <torch/types.h>

#include <ATen/cuda/CUDAApplyUtils.cuh>

#define CHECK_CUDA(x) \
  TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) \
  TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) \
  CHECK_CUDA(x);       \
  CHECK_CONTIGUOUS(x)

namespace {
int const threadsPerBlock = sizeof(unsigned long long) * 8;
}

#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)

template <typename T, typename T_int>
__global__ void dynamic_voxelize_kernel(
    const T* points, T_int* coors, const float voxel_x, const float voxel_y,
    const float voxel_z, const float coors_x_min, const float coors_y_min,
    const float coors_z_min, const float coors_x_max, const float coors_y_max,
    const float coors_z_max, const int grid_x, const int grid_y,
    const int grid_z, const int num_points, const int num_features,
    const int NDim) {
  //   const int index = blockIdx.x * threadsPerBlock + threadIdx.x;
  CUDA_1D_KERNEL_LOOP(index, num_points) {
    // To save some computation
    auto points_offset = points + index * num_features;
    auto coors_offset = coors + index * NDim;
    int c_x = floor((points_offset[0] - coors_x_min) / voxel_x);
    if (c_x < 0 || c_x >= grid_x) {
      coors_offset[0] = -1;
      return;
    }

    int c_y = floor((points_offset[1] - coors_y_min) / voxel_y);
    if (c_y < 0 || c_y >= grid_y) {
      coors_offset[0] = -1;
      coors_offset[1] = -1;
      return;
    }

    int c_z = floor((points_offset[2] - coors_z_min) / voxel_z);
    if (c_z < 0 || c_z >= grid_z) {
      coors_offset[0] = -1;
      coors_offset[1] = -1;
      coors_offset[2] = -1;
    } else {
      coors_offset[0] = c_z;
      coors_offset[1] = c_y;
      coors_offset[2] = c_x;
    }
  }
}

template <typename T, typename T_int>
__global__ void assign_point_to_voxel(const int nthreads, const T* points,
                                      T_int* point_to_voxelidx,
                                      T_int* coor_to_voxelidx, T* voxels,
                                      const int max_points,
                                      const int num_features,
                                      const int num_points, const int NDim) {
  CUDA_1D_KERNEL_LOOP(thread_idx, nthreads) {
    // const int index = blockIdx.x * threadsPerBlock + threadIdx.x;
    int index = thread_idx / num_features;

    int num = point_to_voxelidx[index];
    int voxelidx = coor_to_voxelidx[index];
    if (num > -1 && voxelidx > -1) {
      auto voxels_offset =
          voxels + voxelidx * max_points * num_features + num * num_features;

      int k = thread_idx % num_features;
      voxels_offset[k] = points[thread_idx];
    }
  }
}

template <typename T, typename T_int>
__global__ void assign_voxel_coors(const int nthreads, T_int* coor,
                                   T_int* point_to_voxelidx,
                                   T_int* coor_to_voxelidx, T_int* voxel_coors,
                                   const int num_points, const int NDim) {
  CUDA_1D_KERNEL_LOOP(thread_idx, nthreads) {
    // const int index = blockIdx.x * threadsPerBlock + threadIdx.x;
    // if (index >= num_points) return;
    int index = thread_idx / NDim;
    int num = point_to_voxelidx[index];
    int voxelidx = coor_to_voxelidx[index];
    if (num == 0 && voxelidx > -1) {
      auto coors_offset = voxel_coors + voxelidx * NDim;
      int k = thread_idx % NDim;
      coors_offset[k] = coor[thread_idx];
    }
  }
}

template <typename T_int>
__global__ void point_to_voxelidx_kernel(const T_int* coor,
                                         T_int* point_to_voxelidx,
                                         T_int* point_to_pointidx,
                                         const int max_points,
                                         const int max_voxels,
                                         const int num_points, const int NDim) {
  CUDA_1D_KERNEL_LOOP(index, num_points) {
    auto coor_offset = coor + index * NDim;
    // skip invalid points
    if ((index >= num_points) || (coor_offset[0] == -1)) return;

    int num = 0;
    int coor_x = coor_offset[0];
    int coor_y = coor_offset[1];
    int coor_z = coor_offset[2];
    // only calculate the coors before this coor[index]
    for (int i = 0; i < index; ++i) {
      auto prev_coor = coor + i * NDim;
      if (prev_coor[0] == -1) continue;

      // Find all previous points that have the same coors
      // if find the same coor, record it
      if ((prev_coor[0] == coor_x) && (prev_coor[1] == coor_y) &&
          (prev_coor[2] == coor_z)) {
        num++;
        if (num == 1) {
          // point to the same coor that first show up
          point_to_pointidx[index] = i;
        } else if (num >= max_points) {
          // out of boundary
          return;
        }
      }
    }
    if (num == 0) {
      point_to_pointidx[index] = index;
    }
    if (num < max_points) {
      point_to_voxelidx[index] = num;
    }
  }
}

template <typename T_int>
__global__ void determin_voxel_num(
    // const T_int* coor,
    T_int* num_points_per_voxel, T_int* point_to_voxelidx,
    T_int* point_to_pointidx, T_int* coor_to_voxelidx, T_int* voxel_num,
    const int max_points, const int max_voxels, const int num_points) {
  // only calculate the coors before this coor[index]
  for (int i = 0; i < num_points; ++i) {
    // if (coor[i][0] == -1)
    //    continue;
    int point_pos_in_voxel = point_to_voxelidx[i];
    // record voxel
    if (point_pos_in_voxel == -1) {
      // out of max_points or invalid point
      continue;
    } else if (point_pos_in_voxel == 0) {
      // record new voxel
      int voxelidx = voxel_num[0];
      if (voxel_num[0] >= max_voxels) break;
      voxel_num[0] += 1;
      coor_to_voxelidx[i] = voxelidx;
      num_points_per_voxel[voxelidx] = 1;
    } else {
      int point_idx = point_to_pointidx[i];
      int voxelidx = coor_to_voxelidx[point_idx];
      if (voxelidx != -1) {
        coor_to_voxelidx[i] = voxelidx;
        num_points_per_voxel[voxelidx] += 1;
      }
    }
  }
}

namespace voxelization {

int hard_voxelize_gpu(const at::Tensor& points, at::Tensor& voxels,
                      at::Tensor& coors, at::Tensor& num_points_per_voxel,
                      const std::vector<float> voxel_size,
                      const std::vector<float> coors_range,
                      const int max_points, const int max_voxels,
                      const int NDim = 3) {
  // current version tooks about 0.04s for one frame on cpu
  // check device
  CHECK_INPUT(points);

  at::cuda::CUDAGuard device_guard(points.device());

  const int num_points = points.size(0);
  const int num_features = points.size(1);

  const float voxel_x = voxel_size[0];
  const float voxel_y = voxel_size[1];
  const float voxel_z = voxel_size[2];
  const float coors_x_min = coors_range[0];
  const float coors_y_min = coors_range[1];
  const float coors_z_min = coors_range[2];
  const float coors_x_max = coors_range[3];
  const float coors_y_max = coors_range[4];
  const float coors_z_max = coors_range[5];

  const int grid_x = round((coors_x_max - coors_x_min) / voxel_x);
  const int grid_y = round((coors_y_max - coors_y_min) / voxel_y);
  const int grid_z = round((coors_z_max - coors_z_min) / voxel_z);

  // map points to voxel coors
  at::Tensor temp_coors =
      at::zeros({num_points, NDim}, points.options().dtype(at::kInt));

  dim3 grid(std::min(at::cuda::ATenCeilDiv(num_points, 512), 4096));
  dim3 block(512);

  // 1. link point to corresponding voxel coors
  AT_DISPATCH_ALL_TYPES(
      points.scalar_type(), "hard_voxelize_kernel", ([&] {
        dynamic_voxelize_kernel<scalar_t, int>
            <<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
                points.contiguous().data_ptr<scalar_t>(),
                temp_coors.contiguous().data_ptr<int>(), voxel_x, voxel_y,
                voxel_z, coors_x_min, coors_y_min, coors_z_min, coors_x_max,
                coors_y_max, coors_z_max, grid_x, grid_y, grid_z, num_points,
                num_features, NDim);
      }));
  hipDeviceSynchronize();
  AT_CUDA_CHECK(hipGetLastError());

  // 2. map point to the idx of the corresponding voxel, find duplicate coor
  // create some temporary variables
  auto point_to_pointidx = -at::ones(
      {
          num_points,
      },
      points.options().dtype(at::kInt));
  auto point_to_voxelidx = -at::ones(
      {
          num_points,
      },
      points.options().dtype(at::kInt));

  dim3 map_grid(std::min(at::cuda::ATenCeilDiv(num_points, 512), 4096));
  dim3 map_block(512);
  AT_DISPATCH_ALL_TYPES(
      temp_coors.scalar_type(), "determin_duplicate", ([&] {
        point_to_voxelidx_kernel<int>
            <<<map_grid, map_block, 0, at::cuda::getCurrentCUDAStream()>>>(
                temp_coors.contiguous().data_ptr<int>(),
                point_to_voxelidx.contiguous().data_ptr<int>(),
                point_to_pointidx.contiguous().data_ptr<int>(), max_points,
                max_voxels, num_points, NDim);
      }));
  hipDeviceSynchronize();
  AT_CUDA_CHECK(hipGetLastError());

  // 3. determin voxel num and voxel's coor index
  // make the logic in the CUDA device could accelerate about 10 times
  auto coor_to_voxelidx = -at::ones(
      {
          num_points,
      },
      points.options().dtype(at::kInt));
  auto voxel_num = at::zeros(
      {
          1,
      },
      points.options().dtype(at::kInt));  // must be zero from the begining

  AT_DISPATCH_ALL_TYPES(
      temp_coors.scalar_type(), "determin_duplicate", ([&] {
        determin_voxel_num<int><<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
            num_points_per_voxel.contiguous().data_ptr<int>(),
            point_to_voxelidx.contiguous().data_ptr<int>(),
            point_to_pointidx.contiguous().data_ptr<int>(),
            coor_to_voxelidx.contiguous().data_ptr<int>(),
            voxel_num.contiguous().data_ptr<int>(), max_points, max_voxels,
            num_points);
      }));
  hipDeviceSynchronize();
  AT_CUDA_CHECK(hipGetLastError());

  // 4. copy point features to voxels
  // Step 4 & 5 could be parallel
  auto pts_output_size = num_points * num_features;
  dim3 cp_grid(std::min(at::cuda::ATenCeilDiv(pts_output_size, 512), 4096));
  dim3 cp_block(512);
  AT_DISPATCH_ALL_TYPES(
      points.scalar_type(), "assign_point_to_voxel", ([&] {
        assign_point_to_voxel<float, int>
            <<<cp_grid, cp_block, 0, at::cuda::getCurrentCUDAStream()>>>(
                pts_output_size, points.contiguous().data_ptr<float>(),
                point_to_voxelidx.contiguous().data_ptr<int>(),
                coor_to_voxelidx.contiguous().data_ptr<int>(),
                voxels.contiguous().data_ptr<float>(), max_points, num_features,
                num_points, NDim);
      }));
  //   hipDeviceSynchronize();
  //   AT_CUDA_CHECK(hipGetLastError());

  // 5. copy coors of each voxels
  auto coors_output_size = num_points * NDim;
  dim3 coors_cp_grid(
      std::min(at::cuda::ATenCeilDiv(coors_output_size, 512), 4096));
  dim3 coors_cp_block(512);
  AT_DISPATCH_ALL_TYPES(
      points.scalar_type(), "assign_point_to_voxel", ([&] {
        assign_voxel_coors<float, int><<<coors_cp_grid, coors_cp_block, 0,
                                         at::cuda::getCurrentCUDAStream()>>>(
            coors_output_size, temp_coors.contiguous().data_ptr<int>(),
            point_to_voxelidx.contiguous().data_ptr<int>(),
            coor_to_voxelidx.contiguous().data_ptr<int>(),
            coors.contiguous().data_ptr<int>(), num_points, NDim);
      }));
  hipDeviceSynchronize();
  AT_CUDA_CHECK(hipGetLastError());

  auto voxel_num_cpu = voxel_num.to(at::kCPU);
  int voxel_num_int = voxel_num_cpu.data_ptr<int>()[0];

  return voxel_num_int;
}

void dynamic_voxelize_gpu(const at::Tensor& points, at::Tensor& coors,
                          const std::vector<float> voxel_size,
                          const std::vector<float> coors_range,
                          const int NDim = 3) {
  // current version tooks about 0.04s for one frame on cpu
  // check device
  CHECK_INPUT(points);

  at::cuda::CUDAGuard device_guard(points.device());

  const int num_points = points.size(0);
  const int num_features = points.size(1);

  const float voxel_x = voxel_size[0];
  const float voxel_y = voxel_size[1];
  const float voxel_z = voxel_size[2];
  const float coors_x_min = coors_range[0];
  const float coors_y_min = coors_range[1];
  const float coors_z_min = coors_range[2];
  const float coors_x_max = coors_range[3];
  const float coors_y_max = coors_range[4];
  const float coors_z_max = coors_range[5];

  const int grid_x = round((coors_x_max - coors_x_min) / voxel_x);
  const int grid_y = round((coors_y_max - coors_y_min) / voxel_y);
  const int grid_z = round((coors_z_max - coors_z_min) / voxel_z);

  const int col_blocks = at::cuda::ATenCeilDiv(num_points, threadsPerBlock);
  dim3 blocks(col_blocks);
  dim3 threads(threadsPerBlock);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_ALL_TYPES(points.scalar_type(), "dynamic_voxelize_kernel", [&] {
    dynamic_voxelize_kernel<scalar_t, int><<<blocks, threads, 0, stream>>>(
        points.contiguous().data_ptr<scalar_t>(),
        coors.contiguous().data_ptr<int>(), voxel_x, voxel_y, voxel_z,
        coors_x_min, coors_y_min, coors_z_min, coors_x_max, coors_y_max,
        coors_z_max, grid_x, grid_y, grid_z, num_points, num_features, NDim);
  });
  hipDeviceSynchronize();
  AT_CUDA_CHECK(hipGetLastError());

  return;
}

}  // namespace voxelization
