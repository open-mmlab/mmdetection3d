#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <torch/types.h>

#include <ATen/cuda/CUDAApplyUtils.cuh>

#define CHECK_CUDA(x) \
  TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) \
  TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) \
  CHECK_CUDA(x);       \
  CHECK_CONTIGUOUS(x)

namespace {
int const threadsPerBlock = sizeof(unsigned long long) * 8;
}

#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)

template <typename T, typename T_int>
__global__ void dynamic_voxelize_kernel(
    const T* points, T_int* coors, const float voxel_x, const float voxel_y,
    const float voxel_z, const float coors_x_min, const float coors_y_min,
    const float coors_z_min, const float coors_x_max, const float coors_y_max,
    const float coors_z_max, const int grid_x, const int grid_y,
    const int grid_z, const int num_points, const int num_features,
    const int NDim) {
  //   const int index = blockIdx.x * threadsPerBlock + threadIdx.x;
  CUDA_1D_KERNEL_LOOP(index, num_points) {
    // To save some computation
    auto points_offset = points + index * num_features;
    auto coors_offset = coors + index * NDim;
    int c_x = floor((points_offset[0] - coors_x_min) / voxel_x);
    if (c_x < 0 || c_x >= grid_x) {
      coors_offset[0] = -1;
      return;
    }

    int c_y = floor((points_offset[1] - coors_y_min) / voxel_y);
    if (c_y < 0 || c_y >= grid_y) {
      coors_offset[0] = -1;
      coors_offset[1] = -1;
      return;
    }

    int c_z = floor((points_offset[2] - coors_z_min) / voxel_z);
    if (c_z < 0 || c_z >= grid_z) {
      coors_offset[0] = -1;
      coors_offset[1] = -1;
      coors_offset[2] = -1;
    } else {
      coors_offset[0] = c_z;
      coors_offset[1] = c_y;
      coors_offset[2] = c_x;
    }
  }
}

template <typename T, typename T_int>
__global__ void assign_point_to_voxel(const int nthreads, const T* points,
                                      T_int* point_to_voxelidx,
                                      T_int* coor_to_voxelidx, T* voxels,
                                      const int max_points,
                                      const int num_features,
                                      const int num_points, const int NDim) {
  CUDA_1D_KERNEL_LOOP(thread_idx, nthreads) {
    // const int index = blockIdx.x * threadsPerBlock + threadIdx.x;
    int index = thread_idx / num_features;

    int num = point_to_voxelidx[index];
    int voxelidx = coor_to_voxelidx[index];
    if (num > -1 && voxelidx > -1) {
      auto voxels_offset =
          voxels + voxelidx * max_points * num_features + num * num_features;

      int k = thread_idx % num_features;
      voxels_offset[k] = points[thread_idx];
    }
  }
}

template <typename T, typename T_int>
__global__ void assign_voxel_coors(const int nthreads, T_int* coor,
                                   T_int* point_to_voxelidx,
                                   T_int* coor_to_voxelidx, T_int* voxel_coors,
                                   const int num_points, const int NDim) {
  CUDA_1D_KERNEL_LOOP(thread_idx, nthreads) {
    // const int index = blockIdx.x * threadsPerBlock + threadIdx.x;
    // if (index >= num_points) return;
    int index = thread_idx / NDim;
    int num = point_to_voxelidx[index];
    int voxelidx = coor_to_voxelidx[index];
    if (num == 0 && voxelidx > -1) {
      auto coors_offset = voxel_coors + voxelidx * NDim;
      int k = thread_idx % NDim;
      coors_offset[k] = coor[thread_idx];
    }
  }
}

template <typename T_int>
__global__ void point_to_voxelidx_kernel(const T_int* coor,
                                         T_int* point_to_voxelidx,
                                         T_int* point_to_pointidx,
                                         const int max_points,
                                         const int max_voxels,
                                         const int num_points, const int NDim) {
  CUDA_1D_KERNEL_LOOP(index, num_points) {
    auto coor_offset = coor + index * NDim;
    // skip invalid points
    if ((index >= num_points) || (coor_offset[0] == -1)) return;

    int num = 0;
    int coor_x = coor_offset[0];
    int coor_y = coor_offset[1];
    int coor_z = coor_offset[2];
    // only calculate the coors before this coor[index]
    for (int i = 0; i < index; ++i) {
      auto prev_coor = coor + i * NDim;
      if (prev_coor[0] == -1) continue;

      // Find all previous points that have the same coors
      // if find the same coor, record it
      if ((prev_coor[0] == coor_x) && (prev_coor[1] == coor_y) &&
          (prev_coor[2] == coor_z)) {
        num++;
        if (num == 1) {
          // point to the same coor that first show up
          point_to_pointidx[index] = i;
        } else if (num >= max_points) {
          // out of boundary
          return;
        }
      }
    }
    if (num == 0) {
      point_to_pointidx[index] = index;
    }
    if (num < max_points) {
      point_to_voxelidx[index] = num;
    }
  }
}

template <typename T_int>
__global__ void determin_voxel_num(
    // const T_int* coor,
    T_int* num_points_per_voxel, T_int* point_to_voxelidx,
    T_int* point_to_pointidx, T_int* coor_to_voxelidx, T_int* voxel_num,
    const int max_points, const int max_voxels, const int num_points) {
  // only calculate the coors before this coor[index]
  for (int i = 0; i < num_points; ++i) {
    // if (coor[i][0] == -1)
    //    continue;
    int point_pos_in_voxel = point_to_voxelidx[i];
    // record voxel
    if (point_pos_in_voxel == -1) {
      // out of max_points or invalid point
      continue;
    } else if (point_pos_in_voxel == 0) {
      // record new voxel
      int voxelidx = voxel_num[0];
      if (voxel_num[0] >= max_voxels) continue;
      voxel_num[0] += 1;
      coor_to_voxelidx[i] = voxelidx;
      num_points_per_voxel[voxelidx] = 1;
    } else {
      int point_idx = point_to_pointidx[i];
      int voxelidx = coor_to_voxelidx[point_idx];
      if (voxelidx != -1) {
        coor_to_voxelidx[i] = voxelidx;
        num_points_per_voxel[voxelidx] += 1;
      }
    }
  }
}

__global__ void nondisterministic_get_assign_pos(
    const int nthreads, const int32_t *coors_map, int32_t *pts_id,
    int32_t *coors_count, int32_t *reduce_count, int32_t *coors_order) {
  CUDA_1D_KERNEL_LOOP(thread_idx, nthreads) {
    int coors_idx = coors_map[thread_idx];
    if (coors_idx > -1) {
      int32_t coors_pts_pos = atomicAdd(&reduce_count[coors_idx], 1);
      pts_id[thread_idx] = coors_pts_pos;
      if (coors_pts_pos == 0) {
        coors_order[coors_idx] = atomicAdd(coors_count, 1);
      }
    }
  }
}

template<typename T>
__global__ void nondisterministic_assign_point_voxel(
    const int nthreads, const T *points, const int32_t *coors_map,
    const int32_t *pts_id, const int32_t *coors_in,
    const int32_t *reduce_count, const int32_t *coors_order,
    T *voxels, int32_t *coors, int32_t *pts_count, const int max_voxels,
    const int max_points, const int num_features, const int NDim) {
  CUDA_1D_KERNEL_LOOP(thread_idx, nthreads) {
    int coors_idx = coors_map[thread_idx];
    int coors_pts_pos = pts_id[thread_idx];
    if (coors_idx > -1) {
      int coors_pos = coors_order[coors_idx];
      if (coors_pos < max_voxels && coors_pts_pos < max_points) {
        auto voxels_offset =
            voxels + (coors_pos * max_points + coors_pts_pos) * num_features;
        auto points_offset = points + thread_idx * num_features;
        for (int k = 0; k < num_features; k++) {
          voxels_offset[k] = points_offset[k];
        }
        if (coors_pts_pos == 0) {
          pts_count[coors_pos] = min(reduce_count[coors_idx], max_points);
          auto coors_offset = coors + coors_pos * NDim;
          auto coors_in_offset = coors_in + coors_idx * NDim;
          for (int k = 0; k < NDim; k++) {
            coors_offset[k] = coors_in_offset[k];
          }
        }
      }
    }
  }
}

namespace voxelization {

int hard_voxelize_gpu(const at::Tensor& points, at::Tensor& voxels,
                      at::Tensor& coors, at::Tensor& num_points_per_voxel,
                      const std::vector<float> voxel_size,
                      const std::vector<float> coors_range,
                      const int max_points, const int max_voxels,
                      const int NDim = 3) {
  // current version tooks about 0.04s for one frame on cpu
  // check device
  CHECK_INPUT(points);

  at::cuda::CUDAGuard device_guard(points.device());

  const int num_points = points.size(0);
  const int num_features = points.size(1);

  const float voxel_x = voxel_size[0];
  const float voxel_y = voxel_size[1];
  const float voxel_z = voxel_size[2];
  const float coors_x_min = coors_range[0];
  const float coors_y_min = coors_range[1];
  const float coors_z_min = coors_range[2];
  const float coors_x_max = coors_range[3];
  const float coors_y_max = coors_range[4];
  const float coors_z_max = coors_range[5];

  const int grid_x = round((coors_x_max - coors_x_min) / voxel_x);
  const int grid_y = round((coors_y_max - coors_y_min) / voxel_y);
  const int grid_z = round((coors_z_max - coors_z_min) / voxel_z);

  // map points to voxel coors
  at::Tensor temp_coors =
      at::zeros({num_points, NDim}, points.options().dtype(at::kInt));

  dim3 grid(std::min(at::cuda::ATenCeilDiv(num_points, 512), 4096));
  dim3 block(512);

  // 1. link point to corresponding voxel coors
  AT_DISPATCH_ALL_TYPES(
      points.scalar_type(), "hard_voxelize_kernel", ([&] {
        dynamic_voxelize_kernel<scalar_t, int>
            <<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
                points.contiguous().data_ptr<scalar_t>(),
                temp_coors.contiguous().data_ptr<int>(), voxel_x, voxel_y,
                voxel_z, coors_x_min, coors_y_min, coors_z_min, coors_x_max,
                coors_y_max, coors_z_max, grid_x, grid_y, grid_z, num_points,
                num_features, NDim);
      }));
  hipDeviceSynchronize();
  AT_CUDA_CHECK(hipGetLastError());

  // 2. map point to the idx of the corresponding voxel, find duplicate coor
  // create some temporary variables
  auto point_to_pointidx = -at::ones(
      {
          num_points,
      },
      points.options().dtype(at::kInt));
  auto point_to_voxelidx = -at::ones(
      {
          num_points,
      },
      points.options().dtype(at::kInt));

  dim3 map_grid(std::min(at::cuda::ATenCeilDiv(num_points, 512), 4096));
  dim3 map_block(512);
  AT_DISPATCH_ALL_TYPES(
      temp_coors.scalar_type(), "determin_duplicate", ([&] {
        point_to_voxelidx_kernel<int>
            <<<map_grid, map_block, 0, at::cuda::getCurrentCUDAStream()>>>(
                temp_coors.contiguous().data_ptr<int>(),
                point_to_voxelidx.contiguous().data_ptr<int>(),
                point_to_pointidx.contiguous().data_ptr<int>(), max_points,
                max_voxels, num_points, NDim);
      }));
  hipDeviceSynchronize();
  AT_CUDA_CHECK(hipGetLastError());

  // 3. determine voxel num and voxel's coor index
  // make the logic in the CUDA device could accelerate about 10 times
  auto coor_to_voxelidx = -at::ones(
      {
          num_points,
      },
      points.options().dtype(at::kInt));
  auto voxel_num = at::zeros(
      {
          1,
      },
      points.options().dtype(at::kInt));  // must be zero from the beginning

  AT_DISPATCH_ALL_TYPES(
      temp_coors.scalar_type(), "determin_duplicate", ([&] {
        determin_voxel_num<int><<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
            num_points_per_voxel.contiguous().data_ptr<int>(),
            point_to_voxelidx.contiguous().data_ptr<int>(),
            point_to_pointidx.contiguous().data_ptr<int>(),
            coor_to_voxelidx.contiguous().data_ptr<int>(),
            voxel_num.contiguous().data_ptr<int>(), max_points, max_voxels,
            num_points);
      }));
  hipDeviceSynchronize();
  AT_CUDA_CHECK(hipGetLastError());

  // 4. copy point features to voxels
  // Step 4 & 5 could be parallel
  auto pts_output_size = num_points * num_features;
  dim3 cp_grid(std::min(at::cuda::ATenCeilDiv(pts_output_size, 512), 4096));
  dim3 cp_block(512);
  AT_DISPATCH_ALL_TYPES(
      points.scalar_type(), "assign_point_to_voxel", ([&] {
        assign_point_to_voxel<float, int>
            <<<cp_grid, cp_block, 0, at::cuda::getCurrentCUDAStream()>>>(
                pts_output_size, points.contiguous().data_ptr<float>(),
                point_to_voxelidx.contiguous().data_ptr<int>(),
                coor_to_voxelidx.contiguous().data_ptr<int>(),
                voxels.contiguous().data_ptr<float>(), max_points, num_features,
                num_points, NDim);
      }));
  //   hipDeviceSynchronize();
  //   AT_CUDA_CHECK(hipGetLastError());

  // 5. copy coors of each voxels
  auto coors_output_size = num_points * NDim;
  dim3 coors_cp_grid(
      std::min(at::cuda::ATenCeilDiv(coors_output_size, 512), 4096));
  dim3 coors_cp_block(512);
  AT_DISPATCH_ALL_TYPES(
      points.scalar_type(), "assign_point_to_voxel", ([&] {
        assign_voxel_coors<float, int><<<coors_cp_grid, coors_cp_block, 0,
                                         at::cuda::getCurrentCUDAStream()>>>(
            coors_output_size, temp_coors.contiguous().data_ptr<int>(),
            point_to_voxelidx.contiguous().data_ptr<int>(),
            coor_to_voxelidx.contiguous().data_ptr<int>(),
            coors.contiguous().data_ptr<int>(), num_points, NDim);
      }));
  hipDeviceSynchronize();
  AT_CUDA_CHECK(hipGetLastError());

  auto voxel_num_cpu = voxel_num.to(at::kCPU);
  int voxel_num_int = voxel_num_cpu.data_ptr<int>()[0];

  return voxel_num_int;
}

int nondisterministic_hard_voxelize_gpu(
    const at::Tensor &points, at::Tensor &voxels,
    at::Tensor &coors, at::Tensor &num_points_per_voxel,
    const std::vector<float> voxel_size,
    const std::vector<float> coors_range,
    const int max_points, const int max_voxels,
    const int NDim = 3) {

  CHECK_INPUT(points);

  at::cuda::CUDAGuard device_guard(points.device());

  const int num_points = points.size(0);
  const int num_features = points.size(1);

  if (num_points == 0)
    return 0;

  const float voxel_x = voxel_size[0];
  const float voxel_y = voxel_size[1];
  const float voxel_z = voxel_size[2];
  const float coors_x_min = coors_range[0];
  const float coors_y_min = coors_range[1];
  const float coors_z_min = coors_range[2];
  const float coors_x_max = coors_range[3];
  const float coors_y_max = coors_range[4];
  const float coors_z_max = coors_range[5];

  const int grid_x = round((coors_x_max - coors_x_min) / voxel_x);
  const int grid_y = round((coors_y_max - coors_y_min) / voxel_y);
  const int grid_z = round((coors_z_max - coors_z_min) / voxel_z);

  // map points to voxel coors
  at::Tensor temp_coors =
      at::zeros({num_points, NDim}, points.options().dtype(torch::kInt32));

  dim3 grid(std::min(at::cuda::ATenCeilDiv(num_points, 512), 4096));
  dim3 block(512);

  // 1. link point to corresponding voxel coors
  AT_DISPATCH_ALL_TYPES(
      points.scalar_type(), "hard_voxelize_kernel", ([&] {
    dynamic_voxelize_kernel<scalar_t, int>
    <<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
        points.contiguous().data_ptr<scalar_t>(),
        temp_coors.contiguous().data_ptr<int>(), voxel_x, voxel_y,
        voxel_z, coors_x_min, coors_y_min, coors_z_min, coors_x_max,
        coors_y_max, coors_z_max, grid_x, grid_y, grid_z, num_points,
        num_features, NDim);
  }));

  at::Tensor coors_map;
  at::Tensor coors_count;
  at::Tensor coors_order;
  at::Tensor reduce_count;
  at::Tensor pts_id;

  auto coors_clean = temp_coors.masked_fill(temp_coors.lt(0).any(-1, true), -1);

  std::tie(temp_coors, coors_map, reduce_count) =
      at::unique_dim(coors_clean, 0, true, true, false);

  if (temp_coors.index({0, 0}).lt(0).item<bool>()) {
    // the first element of temp_coors is (-1,-1,-1) and should be removed
    temp_coors = temp_coors.slice(0, 1);
    coors_map = coors_map - 1;
  }

  int num_coors = temp_coors.size(0);
  temp_coors = temp_coors.to(torch::kInt32);
  coors_map = coors_map.to(torch::kInt32);

  coors_count = coors_map.new_zeros(1);
  coors_order = coors_map.new_empty(num_coors);
  reduce_count = coors_map.new_zeros(num_coors);
  pts_id = coors_map.new_zeros(num_points);

  dim3 cp_grid(std::min(at::cuda::ATenCeilDiv(num_points, 512), 4096));
  dim3 cp_block(512);
  AT_DISPATCH_ALL_TYPES(points.scalar_type(), "get_assign_pos", ([&] {
    nondisterministic_get_assign_pos<<<cp_grid, cp_block, 0,
    at::cuda::getCurrentCUDAStream()>>>(
        num_points,
        coors_map.contiguous().data_ptr<int32_t>(),
        pts_id.contiguous().data_ptr<int32_t>(),
        coors_count.contiguous().data_ptr<int32_t>(),
        reduce_count.contiguous().data_ptr<int32_t>(),
        coors_order.contiguous().data_ptr<int32_t>());
  }));

  AT_DISPATCH_ALL_TYPES(
      points.scalar_type(), "assign_point_to_voxel", ([&] {
    nondisterministic_assign_point_voxel<scalar_t>
    <<<cp_grid, cp_block, 0, at::cuda::getCurrentCUDAStream()>>>(
        num_points, points.contiguous().data_ptr<scalar_t>(),
        coors_map.contiguous().data_ptr<int32_t>(),
        pts_id.contiguous().data_ptr<int32_t>(),
        temp_coors.contiguous().data_ptr<int32_t>(),
        reduce_count.contiguous().data_ptr<int32_t>(),
        coors_order.contiguous().data_ptr<int32_t>(),
        voxels.contiguous().data_ptr<scalar_t>(),
        coors.contiguous().data_ptr<int32_t>(),
        num_points_per_voxel.contiguous().data_ptr<int32_t>(),
        max_voxels, max_points,
        num_features, NDim);
  }));
  AT_CUDA_CHECK(hipGetLastError());
  return max_voxels < num_coors ? max_voxels : num_coors;
}

void dynamic_voxelize_gpu(const at::Tensor& points, at::Tensor& coors,
                          const std::vector<float> voxel_size,
                          const std::vector<float> coors_range,
                          const int NDim = 3) {
  // current version tooks about 0.04s for one frame on cpu
  // check device
  CHECK_INPUT(points);

  at::cuda::CUDAGuard device_guard(points.device());

  const int num_points = points.size(0);
  const int num_features = points.size(1);

  const float voxel_x = voxel_size[0];
  const float voxel_y = voxel_size[1];
  const float voxel_z = voxel_size[2];
  const float coors_x_min = coors_range[0];
  const float coors_y_min = coors_range[1];
  const float coors_z_min = coors_range[2];
  const float coors_x_max = coors_range[3];
  const float coors_y_max = coors_range[4];
  const float coors_z_max = coors_range[5];

  const int grid_x = round((coors_x_max - coors_x_min) / voxel_x);
  const int grid_y = round((coors_y_max - coors_y_min) / voxel_y);
  const int grid_z = round((coors_z_max - coors_z_min) / voxel_z);

  const int col_blocks = at::cuda::ATenCeilDiv(num_points, threadsPerBlock);
  dim3 blocks(col_blocks);
  dim3 threads(threadsPerBlock);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_ALL_TYPES(points.scalar_type(), "dynamic_voxelize_kernel", [&] {
    dynamic_voxelize_kernel<scalar_t, int><<<blocks, threads, 0, stream>>>(
        points.contiguous().data_ptr<scalar_t>(),
        coors.contiguous().data_ptr<int>(), voxel_x, voxel_y, voxel_z,
        coors_x_min, coors_y_min, coors_z_min, coors_x_max, coors_y_max,
        coors_z_max, grid_x, grid_y, grid_z, num_points, num_features, NDim);
  });
  hipDeviceSynchronize();
  AT_CUDA_CHECK(hipGetLastError());

  return;
}

}  // namespace voxelization
