// Modified from
// https://github.com/sshaoshuai/Pointnet2.PyTorch/tree/master/pointnet2/src/interpolate_gpu.cu


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define THREADS_PER_BLOCK 256
#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))

__global__ void three_nn_kernel(int b, int n, int m,
                                const float *__restrict__ unknown,
                                const float *__restrict__ known,
                                float *__restrict__ dist2,
                                int *__restrict__ idx) {
  // unknown: (B, N, 3)
  // known: (B, M, 3)
  // output:
  //      dist2: (B, N, 3)
  //      idx: (B, N, 3)

  int bs_idx = blockIdx.y;
  int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (bs_idx >= b || pt_idx >= n) return;

  unknown += bs_idx * n * 3 + pt_idx * 3;
  known += bs_idx * m * 3;
  dist2 += bs_idx * n * 3 + pt_idx * 3;
  idx += bs_idx * n * 3 + pt_idx * 3;

  float ux = unknown[0];
  float uy = unknown[1];
  float uz = unknown[2];

  double best1 = 1e40, best2 = 1e40, best3 = 1e40;
  int besti1 = 0, besti2 = 0, besti3 = 0;
  for (int k = 0; k < m; ++k) {
    float x = known[k * 3 + 0];
    float y = known[k * 3 + 1];
    float z = known[k * 3 + 2];
    float d = (ux - x) * (ux - x) + (uy - y) * (uy - y) + (uz - z) * (uz - z);
    if (d < best1) {
      best3 = best2;
      besti3 = besti2;
      best2 = best1;
      besti2 = besti1;
      best1 = d;
      besti1 = k;
    } else if (d < best2) {
      best3 = best2;
      besti3 = besti2;
      best2 = d;
      besti2 = k;
    } else if (d < best3) {
      best3 = d;
      besti3 = k;
    }
  }
  dist2[0] = best1;
  dist2[1] = best2;
  dist2[2] = best3;
  idx[0] = besti1;
  idx[1] = besti2;
  idx[2] = besti3;
}

void three_nn_kernel_launcher(int b, int n, int m, const float *unknown,
                              const float *known, float *dist2, int *idx,
                              hipStream_t stream) {
  // unknown: (B, N, 3)
  // known: (B, M, 3)
  // output:
  //      dist2: (B, N, 3)
  //      idx: (B, N, 3)

  hipError_t err;
  dim3 blocks(DIVUP(n, THREADS_PER_BLOCK),
              b);  // blockIdx.x(col), blockIdx.y(row)
  dim3 threads(THREADS_PER_BLOCK);

  three_nn_kernel<<<blocks, threads, 0, stream>>>(b, n, m, unknown, known,
                                                  dist2, idx);

  err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
    exit(-1);
  }
}
