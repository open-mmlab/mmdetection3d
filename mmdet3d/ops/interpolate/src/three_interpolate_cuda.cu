// Modified from
// https://github.com/sshaoshuai/Pointnet2.PyTorch/tree/master/pointnet2/src/interpolate_gpu.cu


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define THREADS_PER_BLOCK 256
#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))

__global__ void three_interpolate_kernel(int b, int c, int m, int n,
                                         const float *__restrict__ points,
                                         const int *__restrict__ idx,
                                         const float *__restrict__ weight,
                                         float *__restrict__ out) {
  // points: (B, C, M)
  // idx: (B, N, 3)
  // weight: (B, N, 3)
  // output:
  //      out: (B, C, N)

  int bs_idx = blockIdx.z;
  int c_idx = blockIdx.y;
  int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (bs_idx >= b || c_idx >= c || pt_idx >= n) return;

  weight += bs_idx * n * 3 + pt_idx * 3;
  points += bs_idx * c * m + c_idx * m;
  idx += bs_idx * n * 3 + pt_idx * 3;
  out += bs_idx * c * n + c_idx * n;

  out[pt_idx] = weight[0] * points[idx[0]] + weight[1] * points[idx[1]] +
                weight[2] * points[idx[2]];
}

void three_interpolate_kernel_launcher(int b, int c, int m, int n,
                                       const float *points, const int *idx,
                                       const float *weight, float *out,
                                       hipStream_t stream) {
  // points: (B, C, M)
  // idx: (B, N, 3)
  // weight: (B, N, 3)
  // output:
  //      out: (B, C, N)

  hipError_t err;
  dim3 blocks(DIVUP(n, THREADS_PER_BLOCK), c,
              b);  // blockIdx.x(col), blockIdx.y(row)
  dim3 threads(THREADS_PER_BLOCK);
  three_interpolate_kernel<<<blocks, threads, 0, stream>>>(b, c, m, n, points,
                                                           idx, weight, out);

  err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
    exit(-1);
  }
}

__global__ void three_interpolate_grad_kernel(
    int b, int c, int n, int m, const float *__restrict__ grad_out,
    const int *__restrict__ idx, const float *__restrict__ weight,
    float *__restrict__ grad_points) {
  // grad_out: (B, C, N)
  // weight: (B, N, 3)
  // output:
  //      grad_points: (B, C, M)

  int bs_idx = blockIdx.z;
  int c_idx = blockIdx.y;
  int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (bs_idx >= b || c_idx >= c || pt_idx >= n) return;

  grad_out += bs_idx * c * n + c_idx * n + pt_idx;
  weight += bs_idx * n * 3 + pt_idx * 3;
  grad_points += bs_idx * c * m + c_idx * m;
  idx += bs_idx * n * 3 + pt_idx * 3;

  atomicAdd(grad_points + idx[0], grad_out[0] * weight[0]);
  atomicAdd(grad_points + idx[1], grad_out[0] * weight[1]);
  atomicAdd(grad_points + idx[2], grad_out[0] * weight[2]);
}

void three_interpolate_grad_kernel_launcher(int b, int c, int n, int m,
                                            const float *grad_out,
                                            const int *idx, const float *weight,
                                            float *grad_points,
                                            hipStream_t stream) {
  // grad_out: (B, C, N)
  // weight: (B, N, 3)
  // output:
  //      grad_points: (B, C, M)

  hipError_t err;
  dim3 blocks(DIVUP(n, THREADS_PER_BLOCK), c,
              b);  // blockIdx.x(col), blockIdx.y(row)
  dim3 threads(THREADS_PER_BLOCK);
  three_interpolate_grad_kernel<<<blocks, threads, 0, stream>>>(
      b, c, n, m, grad_out, idx, weight, grad_points);

  err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
    exit(-1);
  }
}
