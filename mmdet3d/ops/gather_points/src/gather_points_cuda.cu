
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define TOTAL_THREADS 1024
#define THREADS_PER_BLOCK 256
#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))

__global__ void gather_points_kernel(int b, int c, int n, int m,
                                     const float *__restrict__ points,
                                     const int *__restrict__ idx,
                                     float *__restrict__ out) {
  // points: (B, C, N)
  // idx: (B, M)
  // output:
  //      out: (B, C, M)

  int bs_idx = blockIdx.z;
  int c_idx = blockIdx.y;
  int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (bs_idx >= b || c_idx >= c || pt_idx >= m) return;

  out += bs_idx * c * m + c_idx * m + pt_idx;
  idx += bs_idx * m + pt_idx;
  points += bs_idx * c * n + c_idx * n;
  out[0] = points[idx[0]];
}

void gather_points_kernel_launcher(int b, int c, int n, int npoints,
                                   const float *points, const int *idx,
                                   float *out, hipStream_t stream) {
  // points: (B, C, N)
  // idx: (B, npoints)
  // output:
  //      out: (B, C, npoints)

  hipError_t err;
  dim3 blocks(DIVUP(npoints, THREADS_PER_BLOCK), c,
              b);  // blockIdx.x(col), blockIdx.y(row)
  dim3 threads(THREADS_PER_BLOCK);

  gather_points_kernel<<<blocks, threads, 0, stream>>>(b, c, n, npoints, points,
                                                       idx, out);

  err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
    exit(-1);
  }
}

__global__ void gather_points_grad_kernel(int b, int c, int n, int m,
                                          const float *__restrict__ grad_out,
                                          const int *__restrict__ idx,
                                          float *__restrict__ grad_points) {
  // grad_out: (B, C, M)
  // idx: (B, M)
  // output:
  //      grad_points: (B, C, N)

  int bs_idx = blockIdx.z;
  int c_idx = blockIdx.y;
  int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (bs_idx >= b || c_idx >= c || pt_idx >= m) return;

  grad_out += bs_idx * c * m + c_idx * m + pt_idx;
  idx += bs_idx * m + pt_idx;
  grad_points += bs_idx * c * n + c_idx * n;

  atomicAdd(grad_points + idx[0], grad_out[0]);
}

void gather_points_grad_kernel_launcher(int b, int c, int n, int npoints,
                                        const float *grad_out, const int *idx,
                                        float *grad_points,
                                        hipStream_t stream) {
  // grad_out: (B, C, npoints)
  // idx: (B, npoints)
  // output:
  //      grad_points: (B, C, N)

  hipError_t err;
  dim3 blocks(DIVUP(npoints, THREADS_PER_BLOCK), c,
              b);  // blockIdx.x(col), blockIdx.y(row)
  dim3 threads(THREADS_PER_BLOCK);

  gather_points_grad_kernel<<<blocks, threads, 0, stream>>>(
      b, c, n, npoints, grad_out, idx, grad_points);

  err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
    exit(-1);
  }
}
