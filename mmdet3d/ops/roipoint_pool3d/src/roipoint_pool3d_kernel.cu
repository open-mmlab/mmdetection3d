/*
Point cloud feature pooling
Written by Shaoshuai Shi
All Rights Reserved 2018.
*/


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

#define THREADS_PER_BLOCK 256
#define DIVUP(m,n) ((m) / (n) + ((m) % (n) > 0))
// #define DEBUG


__device__ inline void lidar_to_local_coords(float shift_x, float shift_y, float rot_angle, float &local_x, float &local_y){
    float cosa = cos(-rot_angle), sina = sin(-rot_angle);
    local_x = shift_x * cosa + shift_y * (-sina);
    local_y = shift_x * sina + shift_y * cosa;
}


__device__ inline int check_pt_in_box3d(const float *pt, const float *box3d, float &local_x, float &local_y){
    // param pt: (x, y, z)
    // param box3d: [x, y, z, dx, dy, dz, heading] (x, y, z) is the box center

    const float MARGIN = 1e-5;
    float x = pt[0], y = pt[1], z = pt[2];
    float cx = box3d[0], cy = box3d[1], cz = box3d[2];
    float dx = box3d[3], dy = box3d[4], dz = box3d[5], rz = box3d[6];

    if (fabsf(z - cz) > dz / 2.0) return 0;
    lidar_to_local_coords(x - cx, y - cy, rz, local_x, local_y);
    float in_flag = (fabs(local_x) < dx / 2.0 + MARGIN) & (fabs(local_y) < dy / 2.0 + MARGIN);
    return in_flag;
}


__global__ void assign_pts_to_box3d(int batch_size, int pts_num, int boxes_num, const float *xyz, const float *boxes3d, int *pts_assign){
    // params xyz: (B, N, 3)
    // params boxes3d: (B, M, 7)
    // params pts_assign: (B, N, M): idx of the corresponding box3d, -1 means background points
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int box_idx = blockIdx.y;
    int bs_idx = blockIdx.z;

    if (pt_idx >= pts_num || box_idx >= boxes_num || bs_idx >= batch_size){
        return;
    }
    int assign_idx = bs_idx * pts_num * boxes_num + pt_idx * boxes_num + box_idx;
    pts_assign[assign_idx] = 0;

    int box_offset = bs_idx * boxes_num * 7 + box_idx * 7;
    int pt_offset = bs_idx * pts_num * 3 + pt_idx * 3;


    float local_x = 0, local_y = 0;
    int cur_in_flag = check_pt_in_box3d(xyz + pt_offset, boxes3d + box_offset, local_x, local_y);
    pts_assign[assign_idx] = cur_in_flag;
    // printf("bs=%d, pt=%d, in=%d\n", bs_idx, pt_idx, pts_assign[bs_idx * pts_num + pt_idx]);
}


__global__ void get_pooled_idx(int batch_size, int pts_num, int boxes_num, int sampled_pts_num,
                               const int *pts_assign, int *pts_idx, int *pooled_empty_flag){
    // params xyz: (B, N, 3)
    // params pts_feature: (B, N, C)
    // params pts_assign: (B, N)
    // params pts_idx: (B, M, 512)
    // params pooled_empty_flag: (B, M)

    int boxes_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (boxes_idx >= boxes_num){
        return;
    }

    int bs_idx = blockIdx.y;

    int cnt = 0;
    for (int k = 0; k < pts_num; k++){
        if (pts_assign[bs_idx * pts_num * boxes_num + k * boxes_num + boxes_idx]){
            if (cnt < sampled_pts_num){
                pts_idx[bs_idx * boxes_num * sampled_pts_num + boxes_idx * sampled_pts_num + cnt] = k;
                cnt++;
            }
            else break;
        }
    }

    if (cnt == 0){
        pooled_empty_flag[bs_idx * boxes_num + boxes_idx] = 1;
    }
    else if (cnt < sampled_pts_num){
        // duplicate same points for sampling
        for (int k = cnt; k < sampled_pts_num; k++){
            int duplicate_idx = k % cnt;
            int base_offset = bs_idx * boxes_num * sampled_pts_num + boxes_idx * sampled_pts_num;
            pts_idx[base_offset + k] = pts_idx[base_offset + duplicate_idx];
        }
    }
}


__global__ void roipool3d_forward(int batch_size, int pts_num, int boxes_num, int feature_in_len, int sampled_pts_num,
                                   const float *xyz, const int *pts_idx, const float *pts_feature,
                                   float *pooled_features, int *pooled_empty_flag){
    // params xyz: (B, N, 3)
    // params pts_idx: (B, M, 512)
    // params pts_feature: (B, N, C)
    // params pooled_features: (B, M, 512, 3+C)
    // params pooled_empty_flag: (B, M)

    int sample_pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int box_idx = blockIdx.y;
    int bs_idx = blockIdx.z;

    if (sample_pt_idx >= sampled_pts_num || box_idx >= boxes_num || bs_idx >= batch_size){
        return;
    }

    if (pooled_empty_flag[bs_idx * boxes_num + box_idx]){
        return;
    }

    int temp_idx = bs_idx * boxes_num * sampled_pts_num + box_idx * sampled_pts_num + sample_pt_idx;
    int src_pt_idx = pts_idx[temp_idx];
    int dst_feature_offset = temp_idx * (3 + feature_in_len);

    for (int j = 0; j < 3; j++)
        pooled_features[dst_feature_offset + j] = xyz[bs_idx * pts_num * 3 + src_pt_idx * 3 + j];

    int src_feature_offset = bs_idx * pts_num * feature_in_len + src_pt_idx * feature_in_len;
    for (int j = 0; j < feature_in_len; j++)
        pooled_features[dst_feature_offset + 3 + j] = pts_feature[src_feature_offset + j];
}


void roipool3dLauncher(int batch_size, int pts_num, int boxes_num, int feature_in_len, int sampled_pts_num,
                       const float *xyz, const float *boxes3d, const float *pts_feature, float *pooled_features, int *pooled_empty_flag){

    // printf("batch_size=%d, pts_num=%d, boxes_num=%d\n", batch_size, pts_num, boxes_num);
    int *pts_assign = NULL;
    hipMalloc(&pts_assign, batch_size * pts_num * boxes_num * sizeof(int));  // (batch_size, N, M)
    // cudaMemset(&pts_assign, -1, batch_size * pts_num * boxes_num * sizeof(int));

    dim3 blocks(DIVUP(pts_num, THREADS_PER_BLOCK), boxes_num, batch_size);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);
    assign_pts_to_box3d<<<blocks, threads>>>(batch_size, pts_num, boxes_num, xyz, boxes3d, pts_assign);

    int *pts_idx = NULL;
    hipMalloc(&pts_idx, batch_size * boxes_num * sampled_pts_num * sizeof(int));  // (batch_size, M, sampled_pts_num)

    dim3 blocks2(DIVUP(boxes_num, THREADS_PER_BLOCK), batch_size);  // blockIdx.x(col), blockIdx.y(row)
    get_pooled_idx<<<blocks2, threads>>>(batch_size, pts_num, boxes_num, sampled_pts_num, pts_assign, pts_idx, pooled_empty_flag);

    dim3 blocks_pool(DIVUP(sampled_pts_num, THREADS_PER_BLOCK), boxes_num, batch_size);
    roipool3d_forward<<<blocks_pool, threads>>>(batch_size, pts_num, boxes_num, feature_in_len, sampled_pts_num,
                                                      xyz, pts_idx, pts_feature, pooled_features, pooled_empty_flag);

    hipFree(pts_assign);
    hipFree(pts_idx);

#ifdef DEBUG
    cudaDeviceSynchronize();  // for using printf in kernel function
#endif
}